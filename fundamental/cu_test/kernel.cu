#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <windows.h>
#include <stdlib.h>
#include <math.h>

#include "../../Cuda_by_example/common/book.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<>
//for __syncthreads()
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)

#include <cstdio>
#include <cstdlib>
#include <hip/device_functions.h>
#include <time.h>
#include<hip/hip_texture_types.h>
#include<>

#include <hiprand/hiprand_kernel.h>

#include <hip/hip_vector_types.h>//used for float2,3,4 etc. type
#include <thrust/device_vector.h>//used for thrust library
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

//#include <thrust/remove.h>
//#include <thrust/execution_policy.h>

//#include <random>
//#include <iomanip>
//#include <algorithm>
//#include <set>
//#include <math_helper.h>
//#include "./cutil_math.h"
//#include <thrust/extrema.h>

//#include <C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\common\inc\hip/hip_runtime_api.h>
//some math function such as cross product,
//in some early edition before CUDA5.0 it is included in  <math_helper.h>
//#include <C:\ProgramData\NVIDIA Corporation\CUDA Samples\v8.0\common\inc\helper_string.h>
#define imin(a,b) (a<b?a:b)
#define N 13
#define ndof 9//the elements number of the matrix for svd decomposition
#define fund_fd 7
#define isnp 8//size parameter in fundatmental model
#define N3 3//size for 3*3 matrix
#define p_core 0.2
#define thresh_sampson 2.0//Sampson Distance: Fundamental Matrix Estimation: A Study of Error Criteria

#define MAX(x,y) ((x)>(y)?(x):(y))
#define MIN(x,y) ((x)<(y)?(x):(y))
#define SIGN(a, b) ((b) >= 0.0 ? fabs(a) : -fabs(a))
using namespace std;

/* open file (from "fp") count the number of data in the file
* Returns number of ints actually read, EOF on end of file, EOF-1 on error*/
int return_datanum(FILE *fp)
{
	int j, n = 0;
	float vals;
	while (!feof(fp)) {
		j = fscanf(fp, "%e", &vals);
		//�� ��:int fscanf(FILE *stream, char *format,[argument...]);
		//����ֵ�����ͣ���ֵ����[argument...]�ĸ���
		if (j == EOF)
			break;
		n += j;
	}

	return n;
}

/* open file (from "fp") count the number of data in the file
* Returns number of rows actually read, EOF on end of file, EOF-1 on error
*/
int return_lineno(FILE *fp)
{
	char ch;
	int lineno = 0;
	while (!feof(fp)) {
		if ((ch = fgetc(fp)) == '\n')
			++lineno;
	}
	return lineno;
}

/* reads (from "fp") "nvals" ints into "vals".
* Returns number of ints actually read, EOF on end of file, EOF-1 on error
*/
int readNInts(FILE *fp, int *vals, int nvals, int matlab_flag)
{
	register int i;
	int n, j;
	float x;

	for (i = n = 0; i<nvals; ++i) {
		j = fscanf(fp, "%e", &x);
		if (matlab_flag)
			*(vals + i) = (int)x - 1;
		else
			*(vals + i) = (int)x;
		if (j == EOF) return EOF;

		if (j != 1 || ferror(fp)) return EOF - 1;

		n += j;
	}

	return n;
}

/* reads (from "fp") "nvals" doubles into "vals".
* Returns number of doubles actually read, EOF on end of file, EOF-1 on error
*/
static int readNDoubles(FILE *fp, float *vals, int nvals, int matlab_flag)
{
	register int i;
	int n, j;

	for (i = n = 0; i<nvals; ++i) {
		j = fscanf(fp, "%ef", vals + i);
		if (matlab_flag)
			*(vals + i) -= 1.0;
		if (j == EOF) return EOF;

		if (j != 1 || ferror(fp)) return EOF - 1;

		n += j;
	}
	//for(i=0;i<nvals;++i)
	//	  printf("%lf",*(vals+i));//����������
	return n;
}

//==============================================
__constant__ int dim_m_I_cn24[7], ch_I0s1[2], ch_I0s2[2], ch_I1s1[2], ch_I1s2[2], I0interchange[2], I1interchange[2], dev_n_round[1];
//dim_m_I_cn24[0] is the number of matches pairs. 
//dim_m_I_cn24[1] is the number of columns of I0, dim_m_I_cn24[2] is the number of columns of I1. They are used for the pointer to index the pixel location linearly
//dim_m_I_cn24[3]=extention(c(n,2)), n is the number of matches pairs.
//Extend cn2 to 2^k for bitonic sort since original bitonic algorithm can only cope with vector with length 2^k.
//dim_m_I_cn24[4]=(int)(log(dim_m_I_cn24[3]) / log(2)) used for bitonic sort.
//dim_m_I_cn24[5] is the half of ext_cn2, used to control bitonic cycle.
//dim_m_I_cn24[6]=c(N,4) is the top c(N,4) small distances of matches pairs produced by c(n,2) sampling after bitonic sort, used to extract matches core.
//ch_I0s1 etc. are temporary variables used in bresenham algorithm

//dev_matches is the matches resulting in image2, which will used as share memory variable.
//dev_frames0x etc. are x and y frame coordinations resulting in image1 and 2 respectively.
//dev_cn2_error is the resulting distance error of cn2 matches.
//dev_ij_in_ind=ind(i,j) is the global index of elements in dev_cn2_error
//the correspondence of ind(i,j) and i, j is ind(i,j)=sum_(k=0)^(i-1)(n-k-1)+j-i-1
//dev_ij_in_ind has two columns using int2 type,
//in which the first column dev_ij_in_ind.x recorded i index in matches
//and the second column dev_ij_in_ind.y recorded j index in matches.
__global__ void extract_parallel_dist_cn2(float *frames0x, float *frames0y, float *frames1x, float *frames1y, float *Image0, float *Image1, float *dev_cn2_error, int2 *dev_ij_in_ind)
{
	unsigned int tid = blockIdx.x;
	for (int i = tid; i <dim_m_I_cn24[0] - 1; i += gridDim.x) {
		int coord_pi_I0_x, coord_pi_I0_y, coord_pi_I1_x, coord_pi_I1_y, I0_x, I0_y;//y is denoted as I0_y
		coord_pi_I0_x = round(frames0x[i]);/*frames0(1,pair_i_I0), x coordinate*/
		coord_pi_I0_y = round(frames0y[i]);/*frames0(2,pair_i_I0), y coordinate*/
		coord_pi_I1_x = round(frames1x[i]);/*frames1(1,pair_i_I1), x coordinate*/
		coord_pi_I1_y = round(frames1y[i]);/*frames1(2,pair_i_I1), y coordinate*/
		I0_x = coord_pi_I0_x;//x is denoted as I0_x
		I0_y = coord_pi_I0_y;//y is denoted as I0_y

		for (int j = threadIdx.x + 1; j <dim_m_I_cn24[0]; j += blockDim.x) {
			if (j > i) {
				//printf("i:%d, j:%d, frames0x[i]:%f ,frames0y[i]:%f, frames1x[i]:%f,frames1y[i]:%f,frames0x[j]:%f ,frames0y[j]:%f, frames1x[j]:%f,frames1y[j]:%f\n", i, j, frames0x[i], frames0y[i], frames1x[i], frames1y[i], frames0x[j], frames0y[j], frames1x[j], frames1y[j]);
				int coord_pj_I0_x, coord_pj_I0_y, coord_pj_I1_x, coord_pj_I1_y, I1_x, I1_y;//y is denoted as I1_y
				int n = 0;
				double  temp = 0.0f;
				coord_pj_I0_x = round(frames0x[j]);/*frames0(1,pair_j_I0), x coordinate*/
				coord_pj_I0_y = round(frames0y[j]);/*frames0(2,pair_j_I0), y coordinate*/
				coord_pj_I1_x = round(frames1x[j]);/*frames1(1,pair_j_I1), x coordinate*/
				coord_pj_I1_y = round(frames1y[j]);/*frames1(2,pair_j_I1), y coordinate */
				I1_x = coord_pi_I1_x;//x is denoted as I1_x
				I1_y = coord_pi_I1_y;//y is denoted as I1_y

									 //                                    (pi_I1_x,pi_I1_y)
									 //                               ___---
									 //                      i  ___---
									 //(pi_I0_x, pi_I0_y)___---
									 //
									 //
									 //(pj_I0_x, pj_I0_y)---___
									 //            ---___  j
									 //                  ---___
									 //                        ---___
									 //                              (pj_I1_x, pj_I1_y)
				int d0x = abs(coord_pj_I0_x - coord_pi_I0_x);//x1-x0
				int d0y = abs(coord_pj_I0_y - coord_pi_I0_y);//y1-y0
				int dl0 = d0x*(d0x > d0y) + d0y*(d0x <= d0y);//dl0 = (d0x>d0y ? d0x : d0y);
				int d1x = abs(coord_pj_I1_x - coord_pi_I1_x);
				int d1y = abs(coord_pj_I1_y - coord_pi_I1_y);
				int dl1 = d1x*(d1x > d1y) + d1y*(d1x <= d1y); //dl1 = (d1x>d1y ? d1x : d1y);
				if (dl0 < 5 || dl1 < 5)
				{
					temp = 1000.0f;
					//continue;
				}
				else
				{
					int I0_dx = d0x;
					int I0_dy = d0y;
					int I1_dx = d1x;
					int I1_dy = d1y;
					int dl, ds;
					int I0p_temp, I1p_temp;
					float I0_cur, I1_cur, I0_next, I1_next, I0_temp, I1_temp, I0_cur_4_diff, I1_cur_4_diff;
					double coe;
					int frag;//, step_count = 0

					d0x = I0_dy * (I0_dy > I0_dx) + I0_dx * (I0_dy <= I0_dx);//if I0_dy > I0_dx, exchange dx and dy, let the line grow in y-direction
					d0y = I0_dx * (I0_dy > I0_dx) + I0_dy * (I0_dy <= I0_dx);
					int I0p = (d0y << 1) - d0x;//ek eq.(7)

					d1x = I1_dy * (I1_dy > I1_dx) + I1_dx * (I1_dy <= I1_dx);
					d1y = I1_dx * (I1_dy > I1_dx) + I1_dy * (I1_dy <= I1_dx);
					int I1p = (d1y << 1) - d1x;//ek eq.(7)

					if (dl0 > dl1)
					{
						dl = dl0;
						ds = dl1;
					}
					else
					{
						dl = dl1;//dl is the longer length between l0 and l1
						ds = dl0;//ds is the shorter length between l0 and l1
					}
					//l = (dl0 > dl1)*dl0 + (dl0 <= dl1)*dl1;
					int sl0 = dl0 < dl1;
					//coe = (dl0 - 1) / (dl1 - 1.0)*sl0 + (dl1 - 1) / (dl0 - 1.0) * (1 - sl0);
					if (sl0)
						coe = ((double)(dl0)) / ((double)(dl1));//eq. (5)
					else
						coe = ((double)(dl1)) / ((double)(dl0));
					//I0_temp = tex2D(Image1, I0_x - 1, I0_y - 1);
					//I1_temp = tex2D(Image2, I1_x - 1, I1_y - 1);
					I0_temp = *(Image0 + I0_y * dim_m_I_cn24[1] + I0_x);// I0 + i*NI0 + j
					I1_temp = *(Image1 + I1_y * dim_m_I_cn24[2] + I1_x);
					I0_cur_4_diff = I0_temp;
					I1_cur_4_diff = I1_temp;
					temp = 0.0f;
					//		temp += fabsf(I0_temp - I1_temp);// (I0_temp - I1_temp)*(I0_temp >= I1_temp) + (I1_temp - I0_temp)*(I0_temp < I1_temp);
					//I0interchange_m = I0interchange[I0_dy > I0_dx];
					dl0 = I0interchange[I0_dy > I0_dx];//x1>x0,temp variable of I0interchange_m, dl0 is used for saving memory of GPU
					dl1 = I1interchange[I1_dy > I1_dx];//y1>y0,temp variable of I1interchange_m
					I0_dx = ch_I0s1[coord_pj_I0_x > coord_pi_I0_x];//x1>x0,temp variable of I0s1
					I0_dy = ch_I0s2[coord_pj_I0_y > coord_pi_I0_y];//y1>y0,temp variable of I0s2
					I1_dx = ch_I1s1[coord_pj_I1_x > coord_pi_I1_x];//temp variable of I1s1
					I1_dy = ch_I1s2[coord_pj_I1_y > coord_pi_I1_y];//temp variable of I1s2
					int et0 = 1;//assistant variable
					int et1 = 1;
					int I0_x0 = I0_x;
					int I0_y0 = I0_y;
					for (n = 1; n < dl; n++)
					{
						if (et0 + 1 - sl0)//update variables in I0 using eq. (6)���I0�ϵ��߶ζ�
						{
							I0_cur = I0_temp;//eq. (6)
							I0_x0 += I0_dx*(1 - dl0);//I0_dx is the denotion gxk in eq. (6). dl0 is the denotion exk in eq. (6)
							I0_y0 += I0_dy *dl0;
							I0_y0 += I0_dy *(1 - dl0)*(I0p >= 0);//I0p is the denotion ek in eq. (6)
							I0_x0 += I0_dx *dl0 * (I0p >= 0);

							I0p_temp = I0p;
							I0p += ((d0y << 1) - (d0x << 1))*(I0p_temp >= 0);
							I0p += (d0y << 1)*(I0p_temp < 0);//update ek
							I0_next = *(Image0 + I0_y0 * dim_m_I_cn24[1] + I0_x0);
							//Image1 is read out in C system, ===========used for debug
							//I0_next = I0[(I0_y0-1)  * dim_m_I[1] + (I0_x0-1) ];//Image1 is read out in matlab system
							//I0_next = tex2D(Image1, I0_x0 - 1, I0_y0 - 1);
							//printf("n=%d, (I0_x,I0_y)=(%d,%d), I0_cur=%f  ", n, I0_x0, I0_y0, I0_next);

						}
						if (et1 + sl0)//���I1�ϵ��߶γ�
						{//update variables in I1 using eq. (6)
							I1_cur = I1_temp;
							I1_x += I1_dx *(1 - dl1);
							I1_y += I1_dy *dl1;
							I1_y += I1_dy *(1 - dl1)*(I1p >= 0);
							I1_x += I1_dx *dl1 *(I1p >= 0);
							I1p_temp = I1p;
							I1p += ((d1y << 1) - (d1x << 1))*(I1p_temp >= 0);
							I1p += (d1y << 1)*(I1p_temp < 0);
							I1_next = Image1[I1_y *dim_m_I_cn24[2] + I1_x];
							//I1_next = tex2D(Image2, I1_x - 1, I1_y - 1);
						}
						//frag =  (int)(n*coe + 0.5) - (int)((n - 1)*coe + 0.5);
						frag = (int)(n*coe) - (int)((n - 1)*coe);//used for pixel growth
																 //l1: * * -> l1_elong * * *
																 //l2: * * *
																 //I(l1_elong(0))=I(l1(0));
																 //I(l1_elong[1])=I(l1([coe*1]-[coe*(1-1)]))=I(l1([2/3]-[0]))=I(l1(0-0))=I(l1(0));
																 //I(l1_elong[2])=I(l1([coe*2]-[coe*(2-1)]))=I(l1([2/3*2]-[2/3]))=I(l1(1-0))=I(l1(1));
																 //l1: * *-> l1_elong * * * * * * * * *
																 //l2: * * * * * * * * *
																 //I(l1_elong(0))=I(l1(0));
																 //frag(n=1)=[coe*1]-[coe*(1-1)]=[2/9]-[0]=0; 
																 //frag(n=2)=[coe*2]-[coe*(2-1)]=[2/9*2]-[2/9]=0;
																 //frag(n=3)=[coe*3]-[coe*(3-1)]=[2/9*3]-[2/9*2]=0;
																 //frag(n=4)=[coe*4]-[coe*(4-1)]=[2/9*4]-[2/9*3]=0;
																 //frag(n=5)=[coe*5]-[coe*(5-1)]=[2/9*5]-[2/9*4]=1;skip 
																 //frag(n=6)=[coe*6]-[coe*(6-1)]=[2/9*6]-[2/9*5]=0; 
																 //frag(n=7)=[coe*7]-[coe*(7-1)]=[2/9*7]-[2/9*6]=1-1=0;
																 //frag(n=8)=[coe*8]-[coe*(8-1)]=[2/9*8]-[2/9*7]=1-1=0;
						if (frag)
						{
							et0 = sl0;
							et1 = 1 - sl0;
						}
						else
						{
							et0 = 1 - sl0;
							et1 = sl0;
						}
						if (sl0 > 0)
						{
							I0_temp = I0_cur*(1 - frag) + I0_next*frag;
							//	temp += fabsf(I0_temp - I1_next);
							I1_temp = I1_next;
						}
						else
						{
							I1_temp = I1_cur*(1 - frag) + I1_next*(frag);
							//	temp += fabsf(I0_next - I1_temp);
							I0_temp = I0_next;
						}
						if (frag)
						{
							temp += fabsf(fabsf(I0_cur_4_diff - I0_next) - fabsf(I1_cur_4_diff - I1_next));
							//	step_count++;
							//	if (dim_m_I_cn24[0]>500)
							//		if (temp / step_count > 0.02)//break unnecessary error calculation
							//			break;
							I1_cur_4_diff = I1_next;
							I0_cur_4_diff = I0_next;
						}

					}/*for (n = 1; n < dl; n++)*/
					 //					temp = temp / dl;
					temp = temp / ds;
				}//else			{
				 //================write back temp to distance vector dev_cn2_error
				I1_y = 0;
				for (tid = 0; tid < i; tid++)
					I1_y += dim_m_I_cn24[0] - 1 - tid;
				I1_y += j - i - 1;
				//atomicExch(dev_cn2_error + I1_y, temp);
				*(dev_cn2_error + I1_y) = temp;
				dev_ij_in_ind[I1_y].x = i;//write back index
				dev_ij_in_ind[I1_y].y = j;
			}//if(j>i){
		}/*for (j = i + 1; j <L; j++) {*/
		 //		__syncthreads();//syncthrize in block
		 //		__threadfence();//syncthrize in grid
	}/*for (i = tid; i <L-1;i+= gridDim.x * blockDim.x) {*/
}/*Do the job*/

 //==================Bitonic sort to choose matches core
__global__ void Bitonic_sort_ex_mc(float *dev_cn2_error, unsigned int *dev_cn2_global_ind, int i, int j)
{
	int coord_pi_I1_x, coord_pi_I1_y, I1_x, I1_y;//y is denoted as I1_y
	unsigned int tid;
	//dim_m_I_cn24[3]=ext_cn2,which is the size of vector to sort. 
	//dim_m_I_cn24[4] = (int)(log(dim_m_I_cn24[3]) / log(2));dim_m_I_cn24[5]=c(N,2)/2
	//dim_m_I_cn24[5] is the half of ext_cn2, used to control bitonic cycle.
	tid = blockIdx.x * blockDim.x + threadIdx.x;
	//	for (tid = blockIdx.x * blockDim.x + threadIdx.x; tid < dim_m_I_cn24[3]; tid += gridDim.x * blockDim.x)
	while (tid < dim_m_I_cn24[5])
	{
		coord_pi_I1_x = (tid / (1 << (j - 1)))*(1 << j) + (tid % (1 << (j - 1)));
		coord_pi_I1_y = (coord_pi_I1_x / (1 << i)) % 2;
		I1_x = (coord_pi_I1_y == 0) ? coord_pi_I1_x : (coord_pi_I1_x + (1 << (j - 1)));
		I1_y = (coord_pi_I1_y == 0) ? (coord_pi_I1_x + (1 << (j - 1))) : coord_pi_I1_x;
		if (dev_cn2_error[I1_y] < dev_cn2_error[I1_x])
		{//write down index to save produce time
		 //Compile kernel code for Compute 2.0 and above only
#if __CUDA_ARCH__ >=200
			atomicExch(&dev_cn2_global_ind[I1_x], atomicExch(&dev_cn2_global_ind[I1_y], dev_cn2_global_ind[I1_x])); //Device functions on the GPU
																													//				atomicMin(&dev_cn2[I1_x], atomicMax(&dev_cn2[I1_y], dev_cn2[I1_x]));
			atomicExch(&dev_cn2_error[I1_x], atomicExch(&dev_cn2_error[I1_y], dev_cn2_error[I1_x]));
#endif


		}
		tid += gridDim.x * blockDim.x;
		//	__syncthreads();
	}//while (tid < dim_m_I_cn24[5])
	__syncthreads();
}/*Do the job*/


 //If RANSAC is implemented on a fundamental matrix, which is conducted by epipolar geometry model, 
 //all the coordinates of the matches should be normalized in the interval [-sqrt(2), sqrt(2)]
 //Input: 
 //n: the number of correspondences
 //framesx, framesy: the x- and y-coordinates of correspondences
 //Output:
 //nrmlz_framesx, nrmlz_framesy: the normalized coordinates
 //T1: normal matrix
__host__ __device__ void normal_samples_fund(int n, float *framesx, float *framesy, float *nrmlz_framesx, float *nrmlz_framesy, double *T1)
{
	int i;
	double muI[2] = { 0,0 };
	double rho1 = 0.0;
	for (i = 0; i < n; i++)
	{
		muI[0] += framesx[i];
		muI[1] += framesy[i];
	}
	muI[0] /= double(n);
	muI[1] /= double(n);
	for (i = 0; i < n; i++)
	{
		nrmlz_framesx[i] = framesx[i] - muI[0];// center x-coordinations of the points
		nrmlz_framesy[i] = framesy[i] - muI[1];// center y-coordinations of the points
		rho1 += sqrt(nrmlz_framesx[i] * nrmlz_framesx[i] + nrmlz_framesy[i] * nrmlz_framesy[i]);
	}
	rho1 /= n;
	rho1 = 1.414213562373095 / rho1;//scale factor
	*T1 = rho1;//Row dominant
	*(T1+1) = 0.0;
	*(T1+2) = -rho1*muI[0];// muIx
	*(T1 + 3) = 0.0;
	*(T1 + 4) = rho1;
	*(T1 + 5) = -rho1*muI[1];// muIy
	*(T1 + 6) = 0.0;
	*(T1 + 7) = 0.0;
	*(T1 + 8) = 1.0;
	for (i = 0; i < n; i++)
	{
		nrmlz_framesx[i] = *T1 * framesx[i] + *(T1 + 2);//normalization
		nrmlz_framesy[i] = *(T1 + 4) * framesy[i] + *(T1 + 5);
	}
	//	cout << "muI:" << muI[0]<< muI[1] << endl;
	//	return muI[0];
}

__host__ __device__ static double PYTHAG(double a, double b)
{
	double at = fabs(a), bt = fabs(b), ct, result;
	if (at > bt) { ct = bt / at; result = at * sqrt(1.0 + ct * ct); }
	else if (bt > 0.0) { ct = at / bt; result = bt * sqrt(1.0 + ct * ct); }
	else result = 0.0;
	return(result);
}

//SVD for thin matrix A=USV'
//A is input and output of U in size of m rows and n columns
//diag is the output of S matrix, stored in memory using a vector
//v is the output of V in size of n*n
//rv1 with the same length of sigular value vector, and almost equals 0
__host__ __device__ int svd(double *A, double *diag, double *v, double *rv1, int m, int n)
//For an arbitrary dimensional matrix m by n, you need to replace ndof and isnp in the following test with m and n, respectively.
//And add "int m, int n" in the function declair which is shown as follows:
//__device__ int svd(double A[ndof][ndof], int m, int n, double *diag, double v[isnp][isnp])
{
	int i, j, k, l;
	double f, h, s;
	double anorm = 0.0, g = 0.0, scale = 0.0;

	for (i = 0; i < n; i++)
	{
		/* left-hand reduction */
		l = i + 1;
		rv1[i] = scale * g;
		g = s = scale = 0.0;
		if (i < m)
		{
			for (k = i; k < m; k++)//array[i][j]=*(array +i*n +j)
								   //scale += fabs(A[k][i]);//A[k][i]=*(A+k*n+i)
				scale += fabs(*(A + k*n + i));
			if (scale)
			{
				for (k = i; k < m; k++)
				{
					*(A + k*n + i) = (*(A + k*n + i) / scale);
					s += (*(A + k*n + i) * *(A + k*n + i));
				}
				f = *(A + i*n + i);
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				*(A + i*n + i) = f - g;
				if (i != n - 1)
				{
					for (j = l; j < n; j++)
					{
						for (s = 0.0, k = i; k < m; k++)
							s += *(A + k*n + i) * *(A + k*n + j);
						f = s / h;
						for (k = i; k < m; k++)
							*(A + k*n + j) += f * *(A + k*n + i);
					}
				}
				for (k = i; k < m; k++)
					*(A + k*n + i) = *(A + k*n + i) * scale;
			}
		}
		diag[i] = scale * g;

		/* right-hand reduction */
		g = s = scale = 0.0;
		if (i < m && i != n - 1)
		{
			for (k = l; k < n; k++)
				scale += fabs(*(A + i*n + k));
			if (scale)
			{
				for (k = l; k < n; k++)
				{
					*(A + i*n + k) = *(A + i*n + k) / scale;
					s += *(A + i*n + k) * *(A + i*n + k);
				}
				f = *(A + i*n + l);
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				*(A + i*n + l) = f - g;
				for (k = l; k < n; k++)
					rv1[k] = *(A + i*n + k) / h;
				if (i != m - 1)
				{
					for (j = l; j < m; j++)
					{
						for (s = 0.0, k = l; k < n; k++)
							s += *(A + j*n + k) * *(A + i*n + k);
						for (k = l; k < n; k++)
							*(A + j*n + k) += s * rv1[k];
					}
				}
				for (k = l; k < n; k++)
					*(A + i*n + k) = *(A + i*n + k) * scale;
			}
		}
		anorm = MAX(anorm, (fabs(diag[i]) + fabs(rv1[i])));
	}

	/* accumulate the right-hand transformation */
	for (i = n - 1; i >= 0; i--)
	{
		if (i < n - 1)
		{
			if (g)
			{
				for (j = l; j < n; j++)
					*(v + j*n + i) = (*(A + i*n + j) / *(A + i*n + l)) / g;
				/* double division to avoid underflow */
				for (j = l; j < n; j++)
				{
					for (s = 0.0, k = l; k < n; k++)
						s += (*(A + i*n + k) * *(v + k*n + j));
					for (k = l; k < n; k++)
						*(v + k*n + j) += s * *(v + k*n + i);
				}
			}
			for (j = l; j < n; j++)
				*(v + i*n + j) = *(v + j*n + i) = 0.0;
		}
		*(v + i*n + i) = 1.0;
		g = rv1[i];
		l = i;
	}

	/* accumulate the left-hand transformation */
	for (i = n - 1; i >= 0; i--)
	{
		l = i + 1;
		g = diag[i];
		if (i < n - 1)
			for (j = l; j < n; j++)
				*(A + i*n + j) = 0.0;
		if (g)
		{
			g = 1.0 / g;
			if (i != n - 1)
			{
				for (j = l; j < n; j++)
				{
					for (s = 0.0, k = l; k < m; k++)
						s += *(A + k*n + i) * *(A + k*n + j);
					f = s / *(A + i*n + i) * g;
					for (k = i; k < m; k++)
						*(A + k*n + j) += (f * *(A + k*n + i));
				}
			}
			for (j = i; j < m; j++)
				*(A + j*n + i) = *(A + j*n + i) * g;
		}
		else
		{
			for (j = i; j < m; j++)
				*(A + j*n + i) = 0.0;
		}
		++(*(A + i*n + i));
	}
	int flag, its, jj, nm;
	double c, x, y, z;
	/* diagonalize the bidiagonal form */
	for (k = n - 1; k >= 0; k--)
	{                             /* loop over singular values */
		for (its = 0; its < 30; its++)
		{                         /* loop over allowed iterations */
			flag = 1;
			for (l = k; l >= 0; l--)
			{                     /* test for splitting */
				nm = l - 1;
				if (fabs(rv1[l]) + anorm == anorm)
				{
					flag = 0;
					break;
				}
				if (fabs(diag[nm]) + anorm == anorm)
					break;
			}
			if (flag)
			{
				c = 0.0;
				s = 1.0;
				for (i = l; i <= k; i++)
				{
					f = s * rv1[i];
					if (fabs(f) + anorm != anorm)
					{
						g = diag[i];
						h = PYTHAG(f, g);
						diag[i] = h;
						h = 1.0 / h;
						c = g * h;
						s = (-f * h);
						for (j = 0; j < m; j++)
						{
							y = *(A + j*n + nm);
							z = *(A + j*n + i);
							*(A + j*n + nm) = y * c + z * s;
							*(A + j*n + i) = z * c - y * s;
						}
					}
				}
			}
			z = diag[k];
			if (l == k)
			{                  /* convergence */
				if (z < 0.0)
				{              /* make singular value nonnegative */
					diag[k] = -z;
					for (j = 0; j < n; j++)
						*(v + j*n + k) = (-*(v + j*n + k));
				}
				break;
			}
			if (its >= 30) {
				free((void*)rv1);
				printf("No convergence after 30,000! iterations \n");
				return(0);
			}

			/* shift from bottom 2 x 2 minor */
			x = diag[l];
			nm = k - 1;
			y = diag[nm];
			g = rv1[nm];
			h = rv1[k];
			f = ((y - z) * (y + z) + (g - h) * (g + h)) / (2.0 * h * y);
			g = PYTHAG(f, 1.0);
			f = ((x - z) * (x + z) + h * ((y / (f + SIGN(g, f))) - h)) / x;

			/* next QR transformation */
			c = s = 1.0;
			for (j = l; j <= nm; j++)
			{
				i = j + 1;
				g = rv1[i];
				y = diag[i];
				h = s * g;
				g = c * g;
				z = PYTHAG(f, h);
				rv1[j] = z;
				c = f / z;
				s = h / z;
				f = x * c + g * s;
				g = g * c - x * s;
				h = y * s;
				y = y * c;
				for (jj = 0; jj < n; jj++)
				{
					x = *(v + jj*n + j);
					z = *(v + jj*n + i);
					*(v + jj*n + j) = x * c + z * s;
					*(v + jj*n + i) = z * c - x * s;
				}
				z = PYTHAG(f, h);
				diag[j] = z;
				if (z)
				{
					z = 1.0 / z;
					c = f * z;
					s = h * z;
				}
				f = (c * g) + (s * y);
				x = (c * y) - (s * g);
				for (jj = 0; jj < m; jj++)
				{
					y = *(A + jj*n + j);
					z = *(A + jj*n + i);
					*(A + jj*n + j) = y * c + z * s;
					*(A + jj*n + i) = z * c - y * s;
				}
			}
			rv1[l] = 0.0;
			rv1[k] = f;
			diag[k] = x;
		}
	}
	return(1);
}


//In order to use thin SVD decompostion method bidiagnal decomposition, input matrix A in size of 8*9 is transformed into 9*8 in this application.
__host__ __device__ void initial_A_fund(double A[fund_fd][ndof], double2 *X1, double2 *X2)
{
	int i;
	for (i = 0; i < fund_fd; i++)
	{
		//input matrix A is in size of 7*9.
		A[i][0] = X1[i].x * X2[i].x; //A[i][0]
		A[i][1] = X1[i].y * X2[i].x;//A[i][1]
		A[i][2] = X2[i].x;
		A[i][3] = X1[i].x * X2[i].y;
		A[i][4] = X1[i].y * X2[i].y;
		A[i][5] = X2[i].y;
		A[i][6] = X1[i].x;
		A[i][7] = X1[i].y;
		A[i][8] = 1.0;
	}
}

//patial QR decomposition. Only R is returned, because Q is useless in fundamental model
__host__ __device__ int partial_QR_decomp79(double A[fund_fd][ndof], double sol[2 * ndof])
{
	double R[fund_fd][ndof], norm_square[ndof], temp, normsquare_vect, vect[fund_fd];
	int pivot[ndof], i, j, k, l, ind, ind_temp;
	for (i = 0; i < ndof; i++)
	{
		pivot[i] = i;//pivot initialization
		norm_square[i] = 0;
		for (j = 0; j < fund_fd; j++)
		{
			norm_square[i] += A[j][i] * A[j][i];
			R[j][i] = A[j][i];
		}
	}
	for (i = 0; i < fund_fd; i++)
	{//processing min(rows, columns) columns for a rectangle matrix
		temp = norm_square[i];
		ind = i;//initialization
		for (j = i + 1; j < ndof; j++)
			if (norm_square[j] > temp)
			{
				ind = j;//the column index with a largest norm
				temp = norm_square[j];
			}
		if (ind != i)
		{
			for (j = 0; j < fund_fd; j++)
			{//exchange pivot dominated columns
				temp = A[j][i];
				A[j][i] = A[j][ind];
				A[j][ind] = temp;
				temp = R[j][i];
				R[j][i] = R[j][ind];
				R[j][ind] = temp;

			}
			//update pivot
			ind_temp = pivot[i];
			pivot[i] = pivot[ind];
			pivot[ind] = ind_temp;
			//update norm
			temp = norm_square[i];
			norm_square[i] = norm_square[ind];
			norm_square[ind] = temp;
		}
		//householder processing on R
		for (j = i; j < fund_fd; j++)
			vect[j] = A[j][i];
		if (vect[i]>0)
			vect[i] = vect[i] + sqrt(norm_square[i]);
		else
			vect[i] = vect[i] - sqrt(norm_square[i]);
		normsquare_vect = 0;
		for (j = i; j < fund_fd; j++)
			normsquare_vect += vect[j] * vect[j];//norm vect
		if (normsquare_vect > 1e-10)//update R under the nonzeros condition
		{
			for (j = i; j<fund_fd; j++)
			{
				for (k = i; k<ndof; k++)
				{
					temp = 0;
					for (l = i; l < fund_fd; l++)
						temp += vect[j] * vect[l] * A[l][k];
					R[j][k] -= 2 / (normsquare_vect)*temp;
					//printf("%f ", R[j][k]);
				}
				//printf("\n");
			}
			for (j = i + 1; j < ndof; j++)
				norm_square[j] -= R[i][j] * R[i][j];
			for (j = i; j < fund_fd; j++)
				for (k = i; k < ndof; k++)
					A[j][k] = R[j][k];
		}
	}

	for (i = 0; i < fund_fd; i++)
		for (j = 0; j < ndof; j++)
			if (i > j)
				R[i][j] = 0;
	// do backsubstitution, resulting R is an upper triangular matrix
	for (k = 1; k <= 2; k++)
	{
		//initialize solution
		for (j = fund_fd; j < ndof; j++)
			sol[pivot[j] + (k - 1)*ndof] = 0;
		sol[pivot[ndof - k] + (k - 1)*ndof] = 1;

		// do backsubstitution
		for (i = fund_fd - 1; i >= 0; i--)
		{
			temp = 0;
			if (R[i][i] == 0.0)
				return -1;
			for (j = i + 1; j<ndof; j++)
				temp += R[i][j] * sol[pivot[j] + (k - 1)*ndof];
			sol[pivot[i] + (k - 1)*ndof] = -temp / R[i][i];
		}
	}
	return 0;
}

__host__ __device__ void makePolynomial(double* A, double* B, double* p)
{
	// calculates polynomial p in x, so that det(xA + (1-x)B) = 0
	// where A,B are [3][3] and p is [4] arrays
	// ** CHANGES B to A-B ***
	// so finally det(A + (x-1) B) = 0 

	*p = -((*(B + 2))*(*(B + 4))*(*(B + 6))) + (*(B + 1))*(*(B + 5))*(*(B + 6)) + (*(B + 2))*(*(B + 3))*(*(B + 7)) -
		(*B)*(*(B + 5))*(*(B + 7)) - (*(B + 1))*(*(B + 3))*(*(B + 8)) + (*B)*(*(B + 4))*(*(B + 8));
	*(p + 1) = -((*(A + 8))*(*(B + 1))*(*(B + 3))) + (*(A + 7))*(*(B + 2))*(*(B + 3)) + (*(A + 8))*(*B)*(*(B + 4)) -
		(*(A + 6))*(*(B + 2))*(*(B + 4)) - (*(A + 7))*(*B)*(*(B + 5)) + (*(A + 6))*(*(B + 1))*(*(B + 5)) +
		(*(A + 5))*(*(B + 1))*(*(B + 6)) - (*(A + 4))*(*(B + 2))*(*(B + 6)) - (*(A + 2))*(*(B + 4))*(*(B + 6)) +
		3 * (*(B + 2))*(*(B + 4))*(*(B + 6)) + (*(A + 1))*(*(B + 5))*(*(B + 6)) - 3 * (*(B + 1))*(*(B + 5))*(*(B + 6)) -
		(*(A + 5))*(*B)*(*(B + 7)) + (*(A + 3))*(*(B + 2))*(*(B + 7)) + (*(A + 2))*(*(B + 3))*(*(B + 7)) -
		3 * (*(B + 2))*(*(B + 3))*(*(B + 7)) - (*A)*(*(B + 5))*(*(B + 7)) + 3 * (*B)*(*(B + 5))*(*(B + 7)) +
		((*(A + 4))*(*B) - (*(A + 3))*(*(B + 1)) - (*(A + 1))*(*(B + 3)) + 3 * (*(B + 1))*(*(B + 3)) + (*A)*(*(B + 4)) -
			3 * (*B)*(*(B + 4)))*(*(B + 8));

	*(p + 2) = -((*(A + 3))*(*(A + 8))*(*(B + 1))) + (*(A + 3))*(*(A + 7))*(*(B + 2)) +
		(*(A + 2))*(*(A + 7))*(*(B + 3)) - (*(A + 1))*(*(A + 8))*(*(B + 3)) + 2 * (*(A + 8))*(*(B + 1))*(*(B + 3)) -
		2 * (*(A + 7))*(*(B + 2))*(*(B + 3)) - (*(A + 2))*(*(A + 6))*(*(B + 4)) + (*A)*(*(A + 8))*(*(B + 4)) -
		2 * (*(A + 8))*(*B)*(*(B + 4)) + 2 * (*(A + 6))*(*(B + 2))*(*(B + 4)) + (*(A + 1))*(*(A + 6))*(*(B + 5)) -
		(*A)*(*(A + 7))*(*(B + 5)) + 2 * (*(A + 7))*(*B)*(*(B + 5)) - 2 * (*(A + 6))*(*(B + 1))*(*(B + 5)) +
		2 * (*(A + 2))*(*(B + 4))*(*(B + 6)) - 3 * (*(B + 2))*(*(B + 4))*(*(B + 6)) - 2 * (*(A + 1))*(*(B + 5))*(*(B + 6)) +
		3 * (*(B + 1))*(*(B + 5))*(*(B + 6)) + (*(A + 2))*(*(A + 3))*(*(B + 7)) - 2 * (*(A + 3))*(*(B + 2))*(*(B + 7)) -
		2 * (*(A + 2))*(*(B + 3))*(*(B + 7)) + 3 * (*(B + 2))*(*(B + 3))*(*(B + 7)) + 2 * (*A)*(*(B + 5))*(*(B + 7)) -
		3 * (*B)*(*(B + 5))*(*(B + 7)) + (*(A + 5))*
		(-((*(A + 7))*(*B)) + (*(A + 6))*(*(B + 1)) + (*(A + 1))*(*(B + 6)) - 2 * (*(B + 1))*(*(B + 6)) -
		(*A)*(*(B + 7)) + 2 * (*B)*(*(B + 7))) +
			(-((*(A + 1))*(*(A + 3))) + 2 * (*(A + 3))*(*(B + 1)) + 2 * (*(A + 1))*(*(B + 3)) - 3 * (*(B + 1))*(*(B + 3)) -
				2 * (*A)*(*(B + 4)) + 3 * (*B)*(*(B + 4)))*(*(B + 8)) +
				(*(A + 4))*((*(A + 8))*(*B) - (*(A + 6))*(*(B + 2)) - (*(A + 2))*(*(B + 6)) + 2 * (*(B + 2))*(*(B + 6)) +
		(*A)*(*(B + 8)) - 2 * (*B)*(*(B + 8)));

	for (unsigned int i = 0; i < 9; ++i)
	{
		B[i] = A[i] - B[i];
	}

	*(p + 3) = -((*(B + 2))*(*(B + 4))*(*(B + 6))) + (*(B + 1))*(*(B + 5))*(*(B + 6)) + (*(B + 2))*(*(B + 3))*(*(B + 7)) -
		(*B)*(*(B + 5))*(*(B + 7)) - (*(B + 1))*(*(B + 3))*(*(B + 8)) + (*B)*(*(B + 4))*(*(B + 8));
} // end makePolynomial

__host__ __device__ double real(double2 xx)
{
	return(xx.x);
}
__host__ __device__ double imag(double2 xx)
{
	return(xx.y);
}

__host__ __device__ double2 cucpow(double2 input, double param)
{
	double thita, r;
	double2 output;
	thita = atan(input.y / (input.x + 0.000000000000000000000000000001));
	r = sqrt(input.x*input.x + input.y*input.y);
	r = pow(r, param);
	output.x = r*cos(param*thita);
	output.y = r*sin(param*thita);
	return(output);
}


__host__ __device__ int cubic_roots(double poly[4], double p3roots[N3])
{
	double2 tem, temp2, temp3, factor_w = { -0.5, 0.866025403784439 }, factor_w2{ -0.5, -0.866025403784439 }, x1, x2, x3;
	double temp;
	int n_sol;
	temp = -*(poly + 1) / (*poly) / 3.0;
	//transform general form to Cardano form: x^3+p*x+q=0, where p is stored in p3roots[0], q stored in p3roots[1]
	p3roots[0] = *(poly + 2) / (*poly) - *(poly + 1) / (*poly) * *(poly + 1) / (*poly) / 3.0;//p
	p3roots[1] = *(poly + 3) / (*poly) - *(poly + 1)**(poly + 2) / (*poly) / (*poly) / 3.0 + 2.0**(poly + 1) / (*poly)**(poly + 1) / (*poly)**(poly + 1) / (*poly) / 27.0;//q
	p3roots[2] = -p3roots[1] * 0.5;//-q/2
	p3roots[0] = p3roots[1] * p3roots[1] * 0.25 + p3roots[0] * p3roots[0] * p3roots[0] / 27.0;//(q/2)^2+(p/3)^3
	if (p3roots[0] > 0)//(q/2)^2+(p/3)^3 is a real number
	{
		tem = { sqrt(p3roots[0]),0 };//sqrt( (q/2)^2 + (p/3)^3 )
		temp2 = { p3roots[2] + real(tem),0 };//the first term of x1 before cubic root:-q/2 + sqrt( (q/2)^2 + (p/3)^3 )
		temp3 = { p3roots[2] - real(tem),0 };//the second term of x1 before cubic root: -q/2 - sqrt( (q/2)^2 + (p/3)^3 )
	}
	else//complex number
	{
		tem = { 0, sqrt(-p3roots[0]) };//temp2=*rv1 + i*temp2.y
		temp2 = { p3roots[2], imag(tem) };
		temp3 = { p3roots[2], -imag(tem) };//temp2=*rv1 - i*temp2.y
	}
	//pow cannot work on negative number
	temp2 = cucpow(temp2, 0.33333333333333333333333333333333333333);//cubic root of temp2 cubic_root( -q/2+sqrt(q^2/4+p^3/27) )
	temp3 = cucpow(temp3, 0.33333333333333333333333333333333333333);//cubic root of temp2 cubic_root( -q/2+sqrt(q^2/4+p^3/27) )

	x1 = { temp + real(temp2) + real(temp3),imag(temp2) + imag(temp3) };//x1
	x2 = { temp + real(factor_w)*real(temp2) - imag(factor_w)*imag(temp2),  real(factor_w)*imag(temp2) + imag(factor_w)*real(temp2) };
	//x2
	x2 = { real(x2) + real(factor_w2)*real(temp3) - imag(factor_w2)*imag(temp3), imag(x2) + real(factor_w2)*imag(temp3) + imag(factor_w2)*real(temp3) };
	x3 = { temp + real(factor_w2)*real(temp2) - imag(factor_w2)*imag(temp2),  real(factor_w2)*imag(temp2) + imag(factor_w2)*real(temp2) };
	//x3 
	x3 = { real(x3) + real(factor_w)*real(temp3) - imag(factor_w)*imag(temp3), imag(x3) + real(factor_w)*imag(temp3) + imag(factor_w)*real(temp3) };

	n_sol = 0;
	if (imag(x1) < 0.00000001&imag(x1) > -0.00000001)
	{
		n_sol++; p3roots[0] = real(x1);
	}//x1,x2,x3 stored in A from A+1 to A+3 depending on the head count at *A
	if (imag(x2) < 0.00000001&imag(x2) > -0.00000001)
	{
		n_sol++; p3roots[1] = real(x2);
	}
	if (imag(x3) < 0.00000001&imag(x3) > -0.00000001)
	{
		n_sol++; p3roots[2] = real(x3);
	}
	return(n_sol);
}

//run for host
void initial_A_fund_host(double *A, double2 *X1, double2 *X2, int m)
{
	int i;
	for (i = 0; i < m; i++)
	{
		*(A + i*ndof) = X1[i].x * X2[i].x; //A[i][0]
		*(A + i*ndof + 1) = X1[i].y * X2[i].x;//A[i][1]
		*(A + i*ndof + 2) = X2[i].x;
		*(A + i*ndof + 3) = X1[i].x * X2[i].y;
		*(A + i*ndof + 4) = X1[i].y * X2[i].y;
		*(A + i*ndof + 5) = X2[i].y;
		*(A + i*ndof + 6) = X1[i].x;
		*(A + i*ndof + 7) = X1[i].y;
		*(A + i*ndof + 8) = 1.0;
	}

}

__device__ void get_nrand(int *mask, int mask_num, int n_matches, int s_d_check)
{
	int s_d, i, j;
	hiprandState localState;
	hiprand_init(clock64(), threadIdx.x, 0, &localState);
	//	mask[0] = 0; mask[1] = 1; mask[2] = 2; mask[3] = 3;
	for (i = 0; i < mask_num; i++)
	{
		//				printf("threadIdx.x:%d, %d, %d\n", threadIdx.x, mask_num, s_d_check);

		while (1)
		{
			s_d = 0;
			for (i = 0; i < mask_num; i++)
			{
				*(mask + i) = hiprand_uniform(&localState) * (n_matches - 1);//x=rand(), results in 0<=x<RAND_MAX
				if (*(mask + i) == n_matches)
					printf("============================================data out!!!!!!!!!!!!!, need to revise 'n_matches' to 'n_matches-1'\n");
			}
			for (i = 0; i < mask_num - 1; i++)
				for (j = i + 1; j < mask_num; j++)
					if (*(mask + i) != *(mask + j))
						s_d++;
			if (s_d == s_d_check)
				break;
		}
	}
}

//Let the minimum singular value be zero
__host__ __device__ void min_diag_0(double *diag3)
{
	double temp;
	int i, mask = 0;
	temp = *diag3;
	for (i = 0; i < N3; i++)
		if (*(diag3 + i) < temp)
		{
			mask = i;
			temp = *(diag3 + i);
		}
	*(diag3 + mask) = 0.0;
}

//F_star=U*diag3*V', U, diag3, V are all in size of 3*3
__host__ __device__ void usvt(double *A1, double *diag3, double *v3, double F_star[N3][N3])
{
	F_star[0][0] = *A1**diag3**v3 + *(A1 + 1)**(diag3 + 1)**(v3 + 1) + *(A1 + 2)**(diag3 + 2)**(v3 + 2);
	F_star[0][1] = *A1**diag3**(v3 + 3) + *(A1 + 1)**(diag3 + 1)**(v3 + 4) + *(A1 + 2)**(diag3 + 2)**(v3 + 5);
	F_star[0][2] = *A1**diag3**(v3 + 6) + *(A1 + 1)**(diag3 + 1)**(v3 + 7) + *(A1 + 2)**(diag3 + 2)**(v3 + 8);
	F_star[1][0] = *(A1 + 3)**diag3**v3 + *(A1 + 4)**(diag3 + 1)**(v3 + 1) + *(A1 + 5)**(diag3 + 2)**(v3 + 2);
	F_star[1][1] = *(A1 + 3)**diag3**(v3 + 3) + *(A1 + 4)**(diag3 + 1)**(v3 + 4) + *(A1 + 5)**(diag3 + 2)**(v3 + 5);
	F_star[1][2] = *(A1 + 3)**diag3**(v3 + 6) + *(A1 + 4)**(diag3 + 1)**(v3 + 7) + *(A1 + 5)**(diag3 + 2)**(v3 + 8);
	F_star[2][0] = *(A1 + 6)**diag3**v3 + *(A1 + 7)**(diag3 + 1)**(v3 + 1) + *(A1 + 8)**(diag3 + 2)**(v3 + 2);
	F_star[2][1] = *(A1 + 6)**diag3**(v3 + 3) + *(A1 + 7)**(diag3 + 1)**(v3 + 4) + *(A1 + 8)**(diag3 + 2)**(v3 + 5);
	F_star[2][2] = *(A1 + 6)**diag3**(v3 + 6) + *(A1 + 7)**(diag3 + 1)**(v3 + 7) + *(A1 + 8)**(diag3 + 2)**(v3 + 8);
}

//count inliers using sampson distances
__host__ __device__ int consensus_check(int n_matches, double F_star[N3][N3], float *frames1x, float *frames1y, float *frames2x, float *frames2y)
{
	int k, inlier_count = 0;
	double Fx1_temp0, Fx1_temp1, Fx1_temp2, x2tFx1_temp, Ftx2_temp0, Ftx2_temp1;
	for (k = 0; k < n_matches; k++)
	{
		Fx1_temp0 = F_star[0][0] * *(frames1x + k) + F_star[0][1] * *(frames1y + k) + F_star[0][2];
		Fx1_temp1 = F_star[1][0] * *(frames1x + k) + F_star[1][1] * *(frames1y + k) + F_star[1][2];
		Fx1_temp2 = F_star[2][0] * *(frames1x + k) + F_star[2][1] * *(frames1y + k) + F_star[2][2];
		x2tFx1_temp = *(frames2x + k)*Fx1_temp0 + *(frames2y + k)*Fx1_temp1 + Fx1_temp2;
		Ftx2_temp0 = F_star[0][0] * *(frames2x + k) + F_star[1][0] * *(frames2y + k) + F_star[2][0];
		Ftx2_temp1 = F_star[0][1] * *(frames2x + k) + F_star[1][1] * *(frames2y + k) + F_star[2][1];
		x2tFx1_temp = x2tFx1_temp*x2tFx1_temp / (Fx1_temp0*Fx1_temp0 + Fx1_temp1*Fx1_temp1 + Ftx2_temp0*Ftx2_temp0 + Ftx2_temp1*Ftx2_temp1);
		if (x2tFx1_temp < thresh_sampson)
		{
			inlier_count++;
		}
	}
	return(inlier_count);
}


//Denormalize to normalization plan
__host__ __device__ void denormalization(double F_star[N3][N3], double T0[ndof], double T1[ndof], double F[N3][N3])
{
	int i, j, k;
	for (i = 0; i < N3; i++)
		for (j = 0; j < N3; j++)
			F[i][j] = 0;
	for (i = 0; i < N3; i++)//T1'*F_star
		for (j = 0; j < N3; j++)
			for (k = 0; k < N3; k++)
				F[i][j] += T1[k*N3+i] * F_star[k][j];
	for (i = 0; i < N3; i++)
		for (j = 0; j < N3; j++)
			F_star[i][j] = F[i][j];
	for (i = 0; i < N3; i++)//T1'*F_star*T0
		for (j = 0; j < N3; j++)
		{
			F[i][j] = 0;
			for (k = 0; k < N3; k++)
				F[i][j] += F_star[i][k] * T0[k*N3 + j];
		}
}

//RANSAC implementation
//(dev_frames0x[X[0]],dev_frames0y[X[0]]) is a sample (x,y)-coordinate in image1, (dev_frames1x[X[0]],dev_frames1y[X[0]]) is a sample (x,y)-coordinate in image2. They are a match pair.
//Output:result_ninlers is a vector with the length gridDim stored in Global memory, in which the i-th element is the maximum number of inliers in the i-th block.
//result_MSS is a vector with the length gridDim*7 involving MSS stored in Global memory, which is in form of (i*7, i*7+1, i*7+2, i*7+3,..., i*7+7) in the i-th block.

//Epipolar geometry model: exploring fundamental matrix
//we use 7-points to estimate fundatmental matrix
__global__ void RANSAC_fund(float *dev_nrmlz_frames0x, float *dev_nrmlz_frames0y, float *dev_nrmlz_frames1x, float *dev_nrmlz_frames1y, float *dev_frames0x, float *dev_frames0y, float *dev_frames1x, float *dev_frames1y, double *T0, double *T1, int l_list, int s_d_check, int *cm_ind, int card_core, int *result_ninliers, int *result_MSS)
{
	int hit_ind[fund_fd], i, j, n_sol, temp_count = 0, k = 0, count = 0;
	double2 X1[fund_fd], X2[fund_fd];

	double A[fund_fd][ndof], sol2[2*ndof], poly[4], *f1, *f2, p3roots[N3];// sol2 is the supplement of U so that[A u9] is a square unitary matrix after SVD decomposition.
	double  F[N3][N3], F_star[N3][N3], F_temp[N3][N3];
	f1 = sol2;
	f2 = sol2 + ndof;

	//inlier_count is in size of blockDim.x, every thread read and write its threadId-th element.
	//The blockDim.x elements in inlier_count are the number of inliers produced by every 8-samples test
	//The blockDim.x*8 elements in inlier_count_7points are the interest of 8-samples index in dev_frames.
	extern __shared__ int inlier_count_7points[];
	unsigned int tid = threadIdx.x;
	unsigned int ind_count = tid;
	inlier_count_7points[ind_count] = 0;
	//thread 0 is in charge of the read and write elements: blockDim.x, blockDim.x + 1, ..., blockDim.x + 7
	//thread 1 is in charge of the read and write elements: blockDim.x+8, blockDim.x + 9, ..., blockDim.x + 15
	for (i = 0; i<fund_fd; i++)
		inlier_count_7points[blockDim.x + fund_fd * ind_count + i] = 0;

	//	if (blockIdx.x == 0)
	//	{
	//		printf("blockIdx.x=%d, tid=%d  inlier_count_Io4si=%d  inlier_count_Io4si[%d]=%d \n ", blockIdx.x, threadIdx.x, inlier_count_Io4si[ind_count], ind_count + blockDim.x, inlier_count_Io4si[ind_count + blockDim.x]);
	//		printf( "dev_n_round[0]=%d, dev_filtered_4samples_list[hit_ind[%d]].x=(%d, %d, %d, %d)\n", dev_n_round[0], tid, dev_filtered_4samples_list[hit_ind[tid]].x, dev_filtered_4samples_list[hit_ind[tid]].y, dev_filtered_4samples_list[hit_ind[tid]].z, dev_filtered_4samples_list[hit_ind[tid]].w);
	//	}
	__syncthreads();//syncthrize in block
	//__threadfence();//syncthrize in grid
	tid = blockIdx.x * blockDim.x + threadIdx.x;
	*result_ninliers = 0;
	//	printf("tid=%d, dev_n_round[0]=%d, gridDim.x=%d, blockDim.x=%d, gridDim.x * blockDim.x=%d\n", tid, dev_n_round[0], gridDim.x, blockDim.x, gridDim.x * blockDim.x);
	while (count < dev_n_round[0])
	{
		//produce 7 different intergers to direct sample, samples are drawn in core, and the consensus are checked in n_matches
		//hit_ind is the output that sampled index of the core
		//		printf("threadIdx.x:%d, %d, %d\n", threadIdx.x, card_core, s_d_check);
		get_nrand(hit_ind, fund_fd, card_core, s_d_check);
		//		cm_ind[0] = 408; cm_ind[1] = 229; cm_ind[2] = 2816; cm_ind[3] = 2164;
		//		cm_ind[4] = 3145; cm_ind[5] = 3095; cm_ind[6] = 284;

		for (i = 0; i<fund_fd; i++)
		{
			//X1[i].x = dev_frames0x[cm_ind[i]];//used for test=========================================
			//X1[i].y = dev_frames0y[cm_ind[i]];//
			//X2[i].x = dev_frames1x[cm_ind[i]];//
			//X2[i].y = dev_frames1y[cm_ind[i]];
			X1[i].x = dev_nrmlz_frames0x[cm_ind[hit_ind[i]]];//extract coordinates based on index
			X1[i].y = dev_nrmlz_frames0y[cm_ind[hit_ind[i]]];//hid_ind points to cm_ind and cm_ind points to frame
			X2[i].x = dev_nrmlz_frames1x[cm_ind[hit_ind[i]]];//extract coordinates based on index
			X2[i].y = dev_nrmlz_frames1y[cm_ind[hit_ind[i]]];
			//			printf("i=%d, X1.x=%f,X1.y=%f,X2.x=%f,X2.y=%f\n", i, X1[i].x, X1[i].y, X2[i].x, X2[i].y);
		}

		//Initialize fundatmental transformation.
		//A is in size of 7X9.
		initial_A_fund(A, X1, X2);

		//The partial QR decomposition ignores the implementation on Q
		//Refers https://www.math.tamu.edu/~fnarc/m660/qr_pivot.html
		//and https://www.irisa.fr/sage/wg-statlin/WORKSHOPS/LEMASSOL05/SLIDES/QR/Guyomarch.pdf
		//and https://www.netlib.org/lapack/lug/node42.html
		//and the paper: Parallelization of the QR Decomposition with Column Pivoting Using Column Cyclic Distribution on Multicore and GPU Processors
		//Input: A_star in size of 7*9
		//Output: sol2 is the solution of Rx=0 with the certain style (...., 1, 0) u9=(...., 0, 1)
		partial_QR_decomp79(A, sol2);
		makePolynomial(f1, f2, poly);
		n_sol = cubic_roots(poly, p3roots);

		for (i = 0; i < n_sol; i++)
		{
			for (j = 0; j < N3; j++)
				for (k = 0; k<N3; k++)
					F_star[j][k] = p3roots[i] * f1[j*N3 + k] + (1 - p3roots[i])*f2[j*N3 + k];
			//         f00 f01 f02
			//    =F=  f10 f11 f12(fundamental matrix F is reshaped from a 9 elements vector in column order)
			//         f20 f21 f22
			denormalization(F_star, T0, T1, F_temp);

		//temp_count is the returned number of consensus points.
			temp_count = consensus_check(l_list, F_temp, dev_frames0x, dev_frames0y, dev_frames1x, dev_frames1y);
			if (temp_count > inlier_count_7points[ind_count])
			{
				inlier_count_7points[ind_count] = temp_count;
				for (j = 0; j < fund_fd; j++)//if the thread with id tid results a better results, update MSS
					inlier_count_7points[blockDim.x + fund_fd * ind_count + j] = cm_ind[hit_ind[j]]; //========================cm_ind[i];for the fixed MSS test
					//if (temp_count > 0.1*dim_m_I_cn24[0])//used for sudden death===========
					//atomicExch(&results[0].x, 1);//used for sudden death===============
			}
		}
		tid += gridDim.x * blockDim.x;
		count++;
	}
	__syncthreads();//syncthrize in block
	if (threadIdx.x == 0)//0th thread is used to write back results in golbal memory.
	{
		for (i = 0; i < blockDim.x; i++)//the 0-th thread collects the best results in shared memory
			if (inlier_count_7points[i] > *(result_ninliers + blockIdx.x))
			{
				//inlier_count[0] = inlier_count[i];
				//for (k = 0; k < isnp; k++)int *result_ninliers, int *result_MSS)
				//inlier_count_7points[k * blockDim.x] = inlier_count_7points[i + k * blockDim.x];
				*(result_ninliers + blockIdx.x) = inlier_count_7points[i];//write result back to global memory
				for (k = 0; k < fund_fd; k++)
					result_MSS[fund_fd * blockIdx.x + k] = inlier_count_7points[blockDim.x + fund_fd * i + k];
			}
		//printf("%d, %d\n", blockIdx.x, *(result_ninliers + blockIdx.x));
	}
}

//host_result_MSS: point to the head of an 7 points MSS
//output:
//CS_temp: if CS_temp[i]=1, the i-th correpondence is a true correspondence
//function value is the number of inliers evaluated with the model modeled by the MSS host_result_MSS
int assembleinliers(int *host_result_MSS, float *nrmlz_frames0x, float *nrmlz_frames0y, float *nrmlz_frames1x, float *nrmlz_frames1y, float *frames0x, float *frames0y, float *frames1x, float *frames1y, double *T0, double *T1, float *host_cn2_error_beforesort, int n_matches, int *CS_temp)
{
	double2 X1[fund_fd], X2[fund_fd];
	int hit_ind[fund_fd], i, j, k = 0, n_sol, temp_count, inlier_count = 0;
	
	double A[fund_fd][ndof], sol2[2 * ndof], poly[4], *f1, *f2, p3roots[N3];//
	f1 = sol2;
	f2 = sol2 + ndof;

	double F[N3][N3], F_star[N3][N3], F_temp[N3][N3];
	double Fx1_temp0, Fx1_temp1, Fx1_temp2, x2tFx1_temp, Ftx2_temp0, Ftx2_temp1;
	//double T0_inv[ndof], T1_inv[ndof];

	//printf("\n*(host_result_MSS + i):\n");
	for (i = 0; i<fund_fd; i++)
	{
		//printf("%d ", *(host_result_MSS + i));
		X1[i].x = nrmlz_frames0x[*(host_result_MSS + i)];//extract coordinates based on index
		X1[i].y = nrmlz_frames0y[*(host_result_MSS + i)];//hid_ind points to cm_ind and cm_ind points to frame
		X2[i].x = nrmlz_frames1x[*(host_result_MSS + i)];//extract coordinates based on index
		X2[i].y = nrmlz_frames1y[*(host_result_MSS + i)];
		//printf("i=%d, X1.x=%f,X1.y=%f,X2.x=%f,X2.y=%f\n", i, X1[i].x, X1[i].y, X2[i].x, X2[i].y);
	}
	//A is in size of 9X8.
	initial_A_fund(A, X1, X2);
	//The partial QR decomposition ignores the implementation on Q
	//Refers https://www.math.tamu.edu/~fnarc/m660/qr_pivot.html
	//and https://www.irisa.fr/sage/wg-statlin/WORKSHOPS/LEMASSOL05/SLIDES/QR/Guyomarch.pdf
	//and https://www.netlib.org/lapack/lug/node42.html
	//and the paper: Parallelization of the QR Decomposition with Column Pivoting Using Column Cyclic Distribution on Multicore and GPU Processors
	//Input: A_star in size of 7*9
	//Output: sol2 is the solution of Rx=0 with the certain style (...., 1, 0) u9=(...., 0, 1)
	partial_QR_decomp79(A, sol2);

	makePolynomial(f1, f2, poly);
	n_sol = cubic_roots(poly, p3roots);

	for (i = 0; i < n_sol; i++)
	{
		for (j = 0; j < N3; j++)
			for (k = 0; k<N3; k++)
				F_star[j][k] = p3roots[i] * f1[j*N3 + k] + (1 - p3roots[i])*f2[j*N3 + k];
		//         f00 f01 f02
		//    =F=  f10 f11 f12(fundamental matrix F is reshaped from a 9 elements vector in column order)
		//         f20 f21 f22
		denormalization(F_star, T0, T1, F_temp);

		//for (j = 0; j < ndof; j++)
		//{
			//T0_inv[j] = 0; T1_inv[j] = 0;
		//}
		//T0_inv[0] = 1 / T0[0]; T0_inv[2] = T0[2] / T0[0]; T0_inv[4] = T0_inv[0]; T0_inv[5] = T0[5] / T0[0]; T0_inv[8] = 1.0;
		//T1_inv[0] = 1 / T1[0]; T1_inv[2] = T1[2] / T1[0]; T1_inv[4] = T1_inv[0]; T1_inv[5] = T1[5] / T1[0]; T1_inv[8] = 1.0;

		//temp_count is the returned number of consensus points.
		//count the number of inliers using the resulting fundamental matrix F_temp
		temp_count = consensus_check(n_matches, F_temp, frames0x, frames0y, frames1x, frames1y);
		if (temp_count > inlier_count)
		{
			inlier_count = temp_count;
			for (j = 0; j < N3; j++)
				for (k = 0; k < N3; k++)
					F[j][k] = F_temp[j][k];
		}
	}
	//prepare for CS_temp
	inlier_count = 0;
	for (k = 0; k < n_matches; k++)
	{
		Fx1_temp0 = F[0][0] * *(frames0x + k) + F[0][1] * *(frames0y + k) + F[0][2];
		Fx1_temp1 = F[1][0] * *(frames0x + k) + F[1][1] * *(frames0y + k) + F[1][2];
		Fx1_temp2 = F[2][0] * *(frames0x + k) + F[2][1] * *(frames0y + k) + F[2][2];
		x2tFx1_temp = *(frames1x + k)*Fx1_temp0 + *(frames1y + k)*Fx1_temp1 + Fx1_temp2;
		Ftx2_temp0 = F[0][0] * *(frames1x + k) + F[1][0] * *(frames1y + k) + F[2][0];
		Ftx2_temp1 = F[0][1] * *(frames1x + k) + F[1][1] * *(frames1y + k) + F[2][1];
		x2tFx1_temp = x2tFx1_temp*x2tFx1_temp / (Fx1_temp0*Fx1_temp0 + Fx1_temp1*Fx1_temp1 + Ftx2_temp0*Ftx2_temp0 + Ftx2_temp1*Ftx2_temp1);
		//if (fabs(x2tFx1_temp) < T_dist & *(candi_matches+k))
		//printf("di:%f\n", x2tFx1_temp);
		if (x2tFx1_temp < thresh_sampson)
		{
			*(CS_temp + k) = 1;
			//*proj_error_temp += x2tFx1_temp;
			inlier_count++;
			//printf("di:%f, %d ", x2tFx1_temp, k);
		}
		else
			*(CS_temp + k) = 0;
	}
	return(inlier_count);
}

void nearest_dist(float *framesx, float *framesy, int N_I_star, int *inliers_candidates, int N_mindist, int *near_ind_img1)
{
	int i, j, k, min_ind;
	double dx, dy, *inliers_dist, temp;
	inliers_dist = (double *)malloc(N_I_star *N_I_star * sizeof(double));
	for (i = 0; i<N_I_star; i++)
		for (j = i + 1; j < N_I_star; j++)
		{
			dx = *(framesx + *(inliers_candidates + i)) - *(framesx + *(inliers_candidates + j));
			dy = *(framesy + *(inliers_candidates + i)) - *(framesy + *(inliers_candidates + j));
			*(inliers_dist + i*N_I_star + j) = dx*dx + dy*dy;
			*(inliers_dist + j*N_I_star + i) = *(inliers_dist + i*N_I_star + j);
		}
	for (i = 0; i < N_I_star; i++)
		*(inliers_dist + i*N_I_star + i) = 0;
///	FILE *fp;///for test
///	fp = fopen("ind_inliers.txt", "w");///for test
///	for (i = 0; i < N_I_star; i++)///for test
///	{///for test
///		fprintf(fp, "%d\n", *(inliers_candidates + i));///for test
///	}///for test
///	fclose(fp);///for test
///	fp = fopen("dist.txt", "w");///for test
///	for (i = 0; i < N_I_star; i++)///for test
///	{///for test
///		for (j = 0; j < N_I_star; j++)///for test
///			fprintf(fp, "%f ", *(inliers_dist + i*N_I_star + j));///for test
///		fprintf(fp, "\n");///for test
///	}///for test
///	fclose(fp);///for test
	for (i = 0; i<N_I_star; i++)
		for (k = 0; k<N_mindist; k++)
		{
			temp = 10000000;
			min_ind = 0;
			for (j = 0; j < N_I_star; j++)
				if (temp > *(inliers_dist + i*N_I_star + j)&i != j)
				{
					temp = *(inliers_dist + i*N_I_star + j);
					min_ind = j;
				}
			*(inliers_dist + i*N_I_star + min_ind) = 10000000;
			*(near_ind_img1 + i*N_mindist + k) = min_ind;
		}
///	fp = fopen("near_ind.txt", "w");///for test
///	for (i = 0; i < N_I_star; i++)///for test
///	{///for test
///		for (j = 0; j < N_mindist; j++)///for test
///			fprintf(fp, "%d ", *(near_ind_img1 + i*N_mindist + j));///for test
///		fprintf(fp, "\n");///for test
///	}///for test
///	fclose(fp);///for test

	free(inliers_dist);
}

//output: partially assigned, CS_temp changes the indexes of outliers to 0, otherwise keep 1
//CS_outlieres_temp: partially assigned, CS_outlieres_temp changes the indexes of outliers to 1, otherwise keep 0
int outliers_detec(float *frames0x, float *frames0y, float *frames1x, float *frames1y, int n_matches, int N_I_star, int *CS_temp, int *CS_outlieres_temp)
{
	int i, j, k, t, temp, N_mindist = 6, deg0, deg1, frames_ind0, frames_ind1;
	int *inliers_candidates, *near_ind_img1, *near_ind_img2;
	inliers_candidates = (int *)malloc(N_I_star * sizeof(int));
	near_ind_img1 = (int *)malloc(N_mindist  *N_I_star * sizeof(int));
	near_ind_img2 = (int *)malloc(N_mindist  *N_I_star * sizeof(int));
	k = 0;
	for (i = 0; i < n_matches; i++)
	{
		*(CS_outlieres_temp + i) = 0;
		if (*(CS_temp + i))
		{
			*(inliers_candidates + k) = i;
			k++;
		}
	}
	nearest_dist(frames0x, frames0y, N_I_star, inliers_candidates, N_mindist, near_ind_img1);
	nearest_dist(frames1x, frames1y, N_I_star, inliers_candidates, N_mindist, near_ind_img2);
	temp = 0;
	for (i = 0; i < N_I_star; i++)
	{
		frames_ind1 = *(inliers_candidates + i);
		k = 0; deg0 = 0; deg1 = 0;
		for (j = 0; j < N_mindist; j++)
		{
			frames_ind0 = *(inliers_candidates + *(near_ind_img1 + i*N_mindist + j));
			if (*(frames0x + frames_ind0) == *(frames0x + frames_ind1)&*(frames0y + frames_ind0) == *(frames0y + frames_ind1))
				deg0++;//count many to one correspondences
		}
		for (t = 0; t < N_mindist; t++)
		{
			frames_ind0 = *(inliers_candidates + *(near_ind_img2 + i*N_mindist + t));
			if (*(frames1x + frames_ind0) == *(frames1x + frames_ind1)&*(frames1y + frames_ind0) == *(frames1y + frames_ind1))
				deg1++;//count many to one correspondences
		}
		if (deg1 > deg0)
			deg0 = deg1;

		for (j = 0; j<N_mindist; j++)
			for (t = 0; t < N_mindist; t++)
				if (*(near_ind_img1 + i*N_mindist + j) == *(near_ind_img2 + i*N_mindist + t))
				{
					k++;//count the same neighbors
					break;
				}
		if (k < N_mindist - 2 | deg0>4)//outliers detected
		{
			*(CS_temp + *(inliers_candidates + i)) = 0;
			*(CS_outlieres_temp + *(inliers_candidates + i)) = 1;
			temp++;
		}
	}
	free(inliers_candidates);
	free(near_ind_img1);
	free(near_ind_img2);
	return(temp);
}


//fitting a model F using the inliers in *CS, where *(CS+i)=1 indicates an inlier, ninliers is the total inliers
//Output: F
void reestimate(float *frames0x, float *frames0y, float *frames1x, float *frames1y, int n_matches, int ninliers, int *CS, double F[N3][N3])
{
	int i, j, k;
	double T0[ndof], T1[ndof];
	float *in_frames0x, *in_frames0y, *in_frames1x, *in_frames1y, *nrmlz_in_frames0x, *nrmlz_in_frames0y, *nrmlz_in_frames1x, *nrmlz_in_frames1y;
	// A=USv^T, v9 is a column of v corresponding to the minimum singular value of in S.
	double *A, diag[ndof], v[ndof*ndof], rv1[ndof], v9[ndof];
	//Fundamental matrix is always a rank-2 matrix, we decompose it to svd, and let a minimum singular value be 0, and compose
	double diag3[N3], v3[N3*N3], rv13[N3], F_star[N3][N3];

	double2 *X1_inliers, *X2_inliers;
	int *ind_inliers;
	ind_inliers = (int *)calloc(ninliers, sizeof(int));
	in_frames0x = (float*)malloc(ninliers * sizeof(float));
	in_frames0y = (float*)malloc(ninliers * sizeof(float));
	in_frames1x = (float*)malloc(ninliers * sizeof(float));
	in_frames1y = (float*)malloc(ninliers * sizeof(float));
	nrmlz_in_frames0x = (float*)malloc(ninliers * sizeof(float));
	nrmlz_in_frames0y = (float*)malloc(ninliers * sizeof(float));
	nrmlz_in_frames1x = (float*)malloc(ninliers * sizeof(float));
	nrmlz_in_frames1y = (float*)malloc(ninliers * sizeof(float));

	X1_inliers = (double2*)malloc(ninliers * sizeof(double2));
	X2_inliers = (double2*)malloc(ninliers * sizeof(double2));

	A = (double*)malloc(ninliers * ndof * sizeof(double));


	j = 0;
	for (i = 0; i < n_matches; i++)
	{
		if (*(CS + i))
		{
			*(ind_inliers + j) = i;
			*(in_frames0x + j) = *(frames0x + i);
			*(in_frames0y + j) = *(frames0y + i);
			*(in_frames1x + j) = *(frames1x + i);
			*(in_frames1y + j) = *(frames1y + i);
			j++;
		}
	}
	normal_samples_fund(ninliers, in_frames0x, in_frames0y, nrmlz_in_frames0x, nrmlz_in_frames0y, T0);
	normal_samples_fund(ninliers, in_frames1x, in_frames1y, nrmlz_in_frames1x, nrmlz_in_frames1y, T1);
	for (i = 0; i<ninliers; i++)
	{
		X1_inliers[i].x = nrmlz_in_frames0x[i];//extract coordinates based on index
		X1_inliers[i].y = nrmlz_in_frames0y[i];
		X2_inliers[i].x = nrmlz_in_frames1x[i];//extract coordinates based on index
		X2_inliers[i].y = nrmlz_in_frames1y[i];
		//printf("i=%d, X1.x=%f,X1.y=%f,X2.x=%f,X2.y=%f\n", i, X1[i].x, X1[i].y, X2[i].x, X2[i].y);
	}
	initial_A_fund_host(A, X1_inliers, X2_inliers, ninliers);

	//implement singular value decomposition with thin type.
	i = svd(A, diag, v, rv1, ninliers, ndof);

	*rv1 = 10000.0;
	for (i = 0; i<ndof; i++)//search for the index of the minimum singular value
		if (*(diag + i) < *rv1)
		{
			*rv1 = *(diag + i);
			k = i;//keep the column index to obtain u9
		}

	for (i = 0; i < ndof; i++)//let u9 be the column of v corresponding to the minimum singular value
		v9[i] = *(v + i*ndof + k);

	//	printf("u9: %f, %f, %f, %f, %f", *v9, *(v9 + 1), *(v9 + 2), *(v9 + 3), *(v9 + 4));

	for (j = 0; j < N3; j++)
		for (i = 0; i < N3; i++)
			//to save thread memory, use A to store our interested 3*3 matrix
			//different from corresponding step in ransac, F is the 
			//v0 v1 v2         f00 f01 f02
			//v3 v4 v5    =F=  f10 f11 f12(there is a transpose from u to fundamental matrix F)
			//v6 v7 v8         f20 f21 f22
			*(A + j*N3 + i) = v9[j*N3 + i];

	i = svd(A, diag3, v3, rv13, N3, N3);
	min_diag_0(diag3);

	//F_star = USV^T (A*diag3*v3^T)
	usvt(A, diag3, v3, F_star);
	//Denormalize to original plan
	denormalization(F_star, T0, T1, F);

	free(ind_inliers);
	free(in_frames0x);
	free(in_frames0y);
	free(in_frames1x);
	free(in_frames1y);
	free(nrmlz_in_frames0x);
	free(nrmlz_in_frames0y);
	free(nrmlz_in_frames1x);
	free(nrmlz_in_frames1y);
	free(X1_inliers);
	free(X2_inliers);
	free(A);
	//return(ninliers);//don't need to return the value
}

//estimate projection error	on all the frames, rather on *CS indicated inliers because of error using the reestimated model F
//output: CS, proj_error, *n_inliers
double est_proj_error(float *frames0x, float *frames0y, float *frames1x, float *frames1y, int n_matches, int *CS, int *CS_outlieres, int *n_inliers, double F[N3][N3])
{
	int i, n_eliminated;
	double *proj_error, mean_proj_error=0.0, rv1[3], x1tfx0, rv2[3];
	proj_error = (double *)malloc(n_matches * sizeof(double));
	*n_inliers = 0;

	for (i = 0; i < n_matches; i++)
	{
		*(CS_outlieres + i) = 0;
		*rv1 = *(frames1x + i)*F[0][0] + *(frames1y + i)*F[1][0] + F[2][0];//x1^t*F(:,0)
		*(rv1 + 1) = *(frames1x + i)*F[0][1] + *(frames1y + i)*F[1][1] + F[2][1];//x1^t*F(:,1)
		*(rv1 + 2) = *(frames1x + i)*F[0][2] + *(frames1y + i)*F[1][2] + F[2][2];//x1^t*F(:,2)
		x1tfx0 = *rv1**(frames0x + i) + *(rv1 + 1)**(frames0y + i) + *(rv1 + 2);//x1^t*F*x0
		*rv2 = *(frames0x + i)*F[0][0] + *(frames0y + i)*F[0][1] + F[0][2];
		*(rv2 + 1) = *(frames0x + i)*F[1][0] + *(frames0y + i)*F[1][1] + F[1][2];
		*(rv2 + 2) = x1tfx0*x1tfx0 / (*rv1**rv1 + *(rv1 + 1)**(rv1 + 1) + *rv2**rv2 + *(rv2 + 1)**(rv2 + 1));
		*(CS + i) = 0;
		if (*(rv2 + 2)<thresh_sampson)
		{
			*(proj_error +i)= *(rv2 + 2);
			*(CS + i) = 1;
			(*n_inliers)++;
			//printf("i:%d, *rv1:%f, 2: %f\n", i, *rv1, proj_error);
		}
	}

	for (i = 0; i < 2; i++)
	{
		n_eliminated = outliers_detec(frames0x, frames0y, frames1x, frames1y, n_matches, *n_inliers, CS, CS_outlieres);
		*n_inliers -= n_eliminated;
	}
	for (i = 0; i < n_matches; i++)
		if (*(CS + i))
			mean_proj_error += *(proj_error + i);
	mean_proj_error = mean_proj_error / *n_inliers;
	return(mean_proj_error);
}


int main(int argc, char ** argv) {
	FILE *fp;
	int *matches1, *matches2;
	/*matches is an array points to the matrix of matched pairs*/
	/*<matches(1,j),matches(2,j)> is a pair of match between image I0 and I1*/
	/*matches(1,j) corresponds to the j-th points in frame1*/
	/*matches(2,j) corresponds to the j-th points in frame2*/
	float *f1x, *f1y, *f2x, *f2y, *frames0x, *frames0y, *frames1x, *frames1y;
	int i, j, temp = 0, n_matches, n_frames0, n_frames1;
	char filename[] = "booksh.txt";//comment this statement if filename receives string from main
	//char *filename;//designed for receiving string from main
	//filename = argv[1];//designed for receiving string from main

	char fn_matches[200] = ".\\kusvod2\\matches_";
	strcat(fn_matches, filename);
	char fn_frames0[200] = ".\\kusvod2\\frames1_";
	strcat(fn_frames0, filename);
	char fn_frames1[200] = ".\\kusvod2\\frames2_";
	strcat(fn_frames1, filename);
	int n_I0, r_I0, c_I0, n_I1, r_I1, c_I1;
	/*n_I0 is the total number of pixels in I0, n_I0=r_I0*c_I0*/
	/*r_I0 is the number of rows of I0*/
	/*c_I0 is the number of column of I0*/
	char fn_I0[200] = ".\\kusvod2\\I1_";
	strcat(fn_I0, filename);
	char fn_I1[200] = ".\\kusvod2\\I2_";
	strcat(fn_I1, filename);

	char file_a[200] = ".\\kusvod2\\csac_results\\csac_inliers_";
	char outliersstr[200] = ".\\kusvod2\\csac_results\\ind_outliers_";
	char file_F[200] = ".\\kusvod2\\csac_results\\F_";
	char file_abstraction[200] = ".\\kusvod2\\csac_results\\abs_";
	char error_fn[200] = ".\\kusvod2\\csac_results\\err_";
	//==============================================
	/*diagnose the size/length of matches*/
	if ((fp = fopen(fn_matches, "r")) == NULL) {
		fprintf(stderr, "error opening file %s!\n", fn_matches);
		exit(1);
	}
	n_matches = return_datanum(fp) / 2;
	if (n_matches > 65000)
	{
		printf("there are too many corresponding matches, which exceed INT_MAX.\n");
		return(0);
	}
	//	printf("n=%d ", n_matches);
	//==============================================

	//==============================================
	/*Read out matches based its length*/
	matches1 = (int *)malloc(n_matches * sizeof(int));
	matches2 = (int *)malloc(n_matches * sizeof(int));
	rewind(fp);
	n_matches = readNInts(fp, matches1, n_matches, 0);
	//////=========================for test
	//1 means the n ints are imported from software matlab, data in matches file is start from 1.
	//when they are used in C, they should substract 1
	n_matches = readNInts(fp, matches2, n_matches, 0);
	fclose(fp);
	//==============================================

	//==============================================
	/*diagnose the size/length of fframes0*/
	if ((fp = fopen(fn_frames0, "r")) == NULL) {
		fprintf(stderr, "error opening file %s!\n", fn_frames0);
		exit(1);
	}
	n_frames0 = return_datanum(fp) / 2;
	//	printf("n=%d ", n_frames0);
	//==============================================

	//==============================================
	/*Read out frames0 based its length*/
	f1x = (float *)malloc(n_frames0 * sizeof(float));
	f1y = (float *)malloc(n_frames0 * sizeof(float));
	rewind(fp);
	n_frames0 = readNDoubles(fp, f1x, n_frames0, 0);
	n_frames0 = readNDoubles(fp, f1y, n_frames0, 0);
	//1 means the n ints are imported from software matlab, data in matches file is start from 1.
	//when they are used in C, the fourth parameter should be set 0
	fclose(fp);
	//==============================================

	//==============================================
	/*diagnose the size/length of fframes1*/
	if ((fp = fopen(fn_frames1, "r")) == NULL) {
		fprintf(stderr, "error opening file %s!\n", fn_frames1);
		exit(1);
	}
	n_frames1 = return_datanum(fp) / 2;
	//==============================================

	//==============================================
	/*Read out frames1 based its length*/
	f2x = (float *)malloc(n_frames1 * sizeof(float));
	f2y = (float *)malloc(n_frames1 * sizeof(float));
	rewind(fp);
	n_frames1 = readNDoubles(fp, f2x, n_frames1, 0);
	n_frames1 = readNDoubles(fp, f2y, n_frames1, 0);
	fclose(fp);

	//rearrange frames data.
	//Before rearrangement, (f1x[matches1[i]], f1y[matches1[i]]) corresponds to point (f2x[matches2[i]], f2y[matches2[i]])
	//After rearrangement, (frames0x[i], frames0y[i]) corresponds to point (frames1x[i], frames1y[i])
	/*( frames1x,frames1y )=( f2x(matches(2,j)),f2y(matches(2,j)) )*/
	/*( frames0x,frames0y )=( f1x(matches(1,j)),f1y(matches(1,j)) )*/
	frames1x = (float *)malloc(n_matches * sizeof(float));
	frames1y = (float *)malloc(n_matches * sizeof(float));
	for (i = 0; i < n_matches; i++)
	{
		*(frames1x + i) = *(f2x + *(matches2 + i));
		*(frames1y + i) = *(f2y + *(matches2 + i));
	}
	frames0x = (float *)malloc(n_matches * sizeof(float));
	frames0y = (float *)malloc(n_matches * sizeof(float));
	for (i = 0; i < n_matches; i++)
	{
		*(frames0x + i) = *(f1x + *(matches1 + i));
		*(frames0y + i) = *(f1y + *(matches1 + i));
	}
	free(f1x);
	free(f1y);
	free(f2x);
	free(f2y);
	//==============================================

	//==============================================
	/*diagnose the size of image I0*/
	if ((fp = fopen(fn_I0, "r")) == NULL) {
		fprintf(stderr, "error opening file %s!\n", fn_I0);
		exit(1);
	}
	n_I0 = return_datanum(fp);
	//	printf("n=%d ", n_I0);
	rewind(fp);
	r_I0 = return_lineno(fp);
	c_I0 = n_I0 / r_I0;
	//	printf("row number ofI0=%d\n", r_I0);
	//	printf("column number ofI0=%d\n", c_I0);
	//==============================================

	//==============================================
	/*Read out I0 based its length*/
	float *I0, *I1;
	I0 = (float *)malloc(n_I0 * sizeof(float));
	rewind(fp);
	n_I0 = readNDoubles(fp, I0, n_I0, 0);
	fclose(fp);
	//==============================================

	//==============================================
	/*diagnose the size of image I1*/
	if ((fp = fopen(fn_I1, "r")) == NULL) {
		fprintf(stderr, "error opening file %s!\n", fn_I1);
		exit(1);
	}
	n_I1 = return_datanum(fp);
	//	printf("n=%d ", n_I1);
	rewind(fp);
	r_I1 = return_lineno(fp);
	c_I1 = n_I1 / r_I1;
	//==============================================

	//==============================================
	/*Read out I1 based its length*/
	I1 = (float *)malloc(n_I1 * sizeof(float));
	rewind(fp);
	n_I1 = readNDoubles(fp, I1, n_I1, 0);
	fclose(fp);
	//==============================================

	//==============================================
	//extract a core of matches
	float *dev_frames0x, *dev_frames1x, *dev_frames0y, *dev_frames1y;
	//dev_frames0x is used for x-coordinates of frames produced in image1. The other three variables have similar sense.
	int cn2, cc4 = N*(N - 1) / 2 * (N - 2) / 3 * (N - 3) / 4;
	if (n_matches % 2)
		cn2 = (n_matches*(n_matches - 1) / 2);
	else
		cn2 = (n_matches / 2 * (n_matches - 1));
	float *host_cn2_error, *host_cn2_error_beforesort, *dev_cn2_error;
	//	host_cn2_error and dev_cn2_error are distance error vector/list/array produced by c(n,2) matches chosen in global matches paires array.
	//before error sorting dev_cn2_error is:
	//e(match0,match1)
	//e(match0,match2)
	//...
	//e(match0,match_(n_matches))
	//e(match1,match2)
	//...
	//e(match1,match_(n_matches))
	//...
	//...
	//e(match_(n_matches-1),match_(n_matches))
	//where matchi is the i-th pair in matches array,
	//matchi(1) and matchi(2) are the coordinates index of match point in frame1 corresponding to image1.
	//e(matchi,matchj) is the intensity distribution mean error between pixes on line1 and line2,
	//line1 is produced by the ith and jth match points on image1,
	//line2 is produced by the ith and jth match points on image2.
	unsigned int *dev_cn2_global_ind, *host_cn2_global_ind;
	//dev_cn2_global_ind and host_cn2_global_ind are indexes of errors before/after sorting corresponds to host_cn2_error and *dev_cn2_error
	//when sorting error, its corresponding index are sorted correspondly.
	//before index sorting dev_cn2_global_ind is:
	//0
	//1
	//2
	//...
	//ext_cn2

	//cc4 is a combinatorial number, which equals c(n,4)
	/*matches is a 2xn array storing one match per column*/
	/*<matches(1,j),matches(2,j)> is a pair of match between image I0 and I1*/
	/*matches(1,j) corresponds to the j-th points in frame1*/
	/*matches(2,j) corresponds to the j-th points in frame2*/
	//frames0 = [X ...
	//         Y ...]
	// The coordination system is in accord with screen coordinate system, such
	//as-------------------------->
	// | O                     X
	// |
	// |
	// | Y
	//   V
	//I0 and I1 are two images normalized in[0, 1]

	//=================Initialize GPU
	hipGetDeviceCount(&i);
	if (i == 0) {	//there is no corresponding GPU used for application.
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	struct hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, 0);
	int blocksPerGrid = device_prop.multiProcessorCount << 1;//imin(32, (ncore + threadsPerBlock - 1) / threadsPerBlock);
	int threadsPerBlock = 256;//based the number of variables used in CUDA __global__ function (total number of registers per block supported by 1080 is 64k)
	printf("blocksPerGrid=%d\n", blocksPerGrid);
	printf("blocksPerGrid=%d\n", threadsPerBlock);

	//============================constant memory initialization.
	//Extend cn2 to 2^k for bitonic sort since original bitonic algorithm can only cope with vector with length 2^k.
	int ext_cn2, half_cn2;
	int temp_ch_I0s1[2] = { -1,1 }, temp_ch_I0s2[2] = { -1,1 }, temp_ch_I1s1[2] = { -1,1 }, temp_ch_I1s2[2] = { -1,1 },
		temp_dim_m_I_cn24[7], temp_I0interchange[2] = { 0,1 }, temp_I1interchange[2] = { 0,1 };
	temp_dim_m_I_cn24[0] = n_matches;
	temp_dim_m_I_cn24[1] = c_I0;//columns number of I0
	temp_dim_m_I_cn24[2] = c_I1;//columns number of I1
	for (i = 0; i < 40; i++)//Initialization of ext_cn2, for parallel Bitonic sort, ext_cn2 should be 2^i, and greater than cn2
	{
		ext_cn2 = pow(2.0, i);
		if (ext_cn2 >= cn2)
			break;
	}
	half_cn2 = ext_cn2 >> 1;
	temp_dim_m_I_cn24[3] = ext_cn2;
	temp_dim_m_I_cn24[4] = (int)(log(ext_cn2) / log(2));//the function of the number of matches c(n,2), temp_dim_m_I_cn24[4] = (int)(log(cn2) / log(2)) which is used in bitonic sort.
	temp_dim_m_I_cn24[5] = half_cn2;//the half of ext_cn2, used to control bitonic cycle.
	temp_dim_m_I_cn24[6] = cc4;//c(n,4)
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dim_m_I_cn24), temp_dim_m_I_cn24, sizeof(int) * 7));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ch_I0s1), temp_ch_I0s1, sizeof(int) * 2));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ch_I0s2), temp_ch_I0s2, sizeof(int) * 2));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ch_I1s1), temp_ch_I1s1, sizeof(int) * 2));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ch_I1s2), temp_ch_I1s2, sizeof(int) * 2));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(I0interchange), temp_I0interchange, sizeof(int) * 2));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(I1interchange), temp_I1interchange, sizeof(int) * 2));

	//=================Initialize variable
	host_cn2_error = (float *)malloc(ext_cn2 * sizeof(float));
	host_cn2_error_beforesort = (float *)malloc(ext_cn2 * sizeof(float));
	host_cn2_global_ind = (unsigned int *)malloc(ext_cn2 * sizeof(unsigned int));
	for (i = 0; i < ext_cn2; i++)
	{
		if (i < cn2)
			*(host_cn2_error + i) = 0;
		else
			*(host_cn2_error + i) = 10000.0;
		*(host_cn2_global_ind + i) = i;
	}

	//=================capture the start time
	//dev_ij_in_ind=ind(i,j) is the global index of elements in dev_cn2_error
	//the correspondence of ind(i,j) and i, j is ind(i,j)=sum_(k=0)^(i-1)(n-k-1)+j-i-1
	//dev_ij_in_ind has two columns using int2 type,
	//in which the first column dev_ij_in_ind.x recorded i index in matches
	//and the second column dev_ij_in_ind.y recorded j index in matches.
	int2 *dev_ij_in_ind, *host_ij_in_ind;
	host_ij_in_ind = (int2 *)malloc(ext_cn2 * sizeof(int2));

	hipEvent_t start1, stop1, start2, stop2;//time counting
	float elapsedTime1, elapsedTime2;//time counting
	float *dev_I0, *dev_I1;
	//	clock_t start, finish;
	double duration;
	_LARGE_INTEGER time_start;  //��ʼʱ��  
	_LARGE_INTEGER time_over;   //����ʱ��  
	double dqFreq;      //��ʱ��Ƶ��  
	LARGE_INTEGER f;    //��ʱ��Ƶ��  
	QueryPerformanceFrequency(&f);
	dqFreq = (double)f.QuadPart;
	QueryPerformanceCounter(&time_start);   //��ʱ��ʼ  
											//	start = clock();
	HANDLE_ERROR(hipEventCreate(&start1));//time counting
	HANDLE_ERROR(hipEventCreate(&stop1));//time counting
	HANDLE_ERROR(hipEventRecord(start1, 0));//time counting
	HANDLE_ERROR(hipMalloc((void**)&dev_frames0x, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_frames1x, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_frames0y, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_frames1y, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_I0, r_I0 * c_I0 * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_I1, r_I1 * c_I1 * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_cn2_error, ext_cn2 * sizeof(float)));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_frames0x, n_frames0 * sizeof(float), hipHostMallocDefault));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_frames1x, n_frames1 * sizeof(float), hipHostMallocDefault));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_frames0y, n_frames0 * sizeof(float), hipHostMallocDefault));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_frames1y, n_frames1 * sizeof(float), hipHostMallocDefault));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_I0, r_I0 * c_I0 * sizeof(float), hipHostMallocDefault));
	//	HANDLE_ERROR(hipHostAlloc((void**)&dev_I1, r_I1 * c_I1 * sizeof(float), hipHostMallocDefault));  
	HANDLE_ERROR(hipMemcpy(dev_frames0x, frames0x, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_frames1x, frames1x, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_frames0y, frames0y, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_frames1y, frames1y, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_I0, I0, r_I0 * c_I0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_I1, I1, r_I1 * c_I1 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_cn2_error, host_cn2_error, ext_cn2 * sizeof(float), hipMemcpyHostToDevice));
	//	HANDLE_ERROR(hipMemcpy(dev_I0, I0, r_I0 * c_I0 * sizeof(float), hipMemcpyHostToDevice));
	//	HANDLE_ERROR(hipMemcpy(dev_I1, I1, r_I1 * c_I1 * sizeof(float), hipMemcpyHostToDevice));

	//dev_matches is the matches resulting in image2, which will used as share memory variable.
	//dev_frames0x etc. are x and y frame coordinations resulting in image1 and 2 respectively.
	//dev_cn2_error is the resulting distance error of cn2 matches.
	//dev_ij_in_ind=ind(i,j) is the retured global index of elements in dev_cn2_error
	//the correspondence of ind(i,j) and i, j is ind(i,j)=sum_(k=0)^(i-1)(n-k-1)+j-i-1
	//dev_ij_in_ind has two columns using int2 type,
	//in which the first column dev_ij_in_ind.x recorded i index in matches
	//and the second column dev_ij_in_ind.y recorded j index in matches.
	HANDLE_ERROR(hipMalloc((void**)&dev_ij_in_ind, ext_cn2 * sizeof(int2)));
	extract_parallel_dist_cn2 << <blocksPerGrid, threadsPerBlock >> >(dev_frames0x, dev_frames0y, dev_frames1x, dev_frames1y, dev_I0, dev_I1, dev_cn2_error, dev_ij_in_ind);
	HANDLE_ERROR(hipMemcpy(host_cn2_error_beforesort, dev_cn2_error, ext_cn2 * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(host_cn2_error, dev_cn2_error, ext_cn2 * sizeof(float), hipMemcpyDeviceToHost));
	//copy dev_ij_in_ind to host_ij_in_ind to prepare for interception core index in global matches array using host CPU.
	HANDLE_ERROR(hipMemcpy(host_ij_in_ind, dev_ij_in_ind, ext_cn2 * sizeof(int2), hipMemcpyDeviceToHost));

	//	fp = fopen("error_ind.txt", "w");
	//	for (i = 0; i < ext_cn2; i++)
	//		fprintf(fp,"%f %d %d\n", host_cn2_error_beforesort[i], host_ij_in_ind[i].x, host_ij_in_ind[i].y);
	//	fclose(fp);

	HANDLE_ERROR(hipEventRecord(stop1, 0));//time counting
	HANDLE_ERROR(hipEventSynchronize(stop1));//time counting
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime1, start1, stop1));//time counting
	printf("Time for calculate priority using two pairs sampling in matches: %3.5f ms\n", elapsedTime1);//time counting
	HANDLE_ERROR(hipFree(dev_ij_in_ind));
	HANDLE_ERROR(hipFree(dev_I0));
	HANDLE_ERROR(hipFree(dev_I1));
	HANDLE_ERROR(hipEventDestroy(start1));
	HANDLE_ERROR(hipEventDestroy(stop1));
	//		for (i = 0; i < ext_cn2; i++)
	//			printf("host_cn2_error: %f %f", *(host_cn2_error + ext_cn2/20), *(host_cn2_error + ext_cn2 / 10));

	//==============================================Bitonic sort dist (dev_cn2_error) to cut top cn4 elements as matches core.
	int *host_cc4_core_global_ind;
	//host_cc4_core_global_ind is intercepted c(n,4) global index corresponds to top min error e(matchi,matchj).
	host_cc4_core_global_ind = (int *)malloc(cc4 * sizeof(unsigned int));
	printf("cn2:%d, ext_cn2:%d", cn2, ext_cn2);
	//	for (i = 0; i < cc4; i++)
	//		*(host_cc4_core_global_ind + i) = 0;
	HANDLE_ERROR(hipEventCreate(&start2));//time counting
	HANDLE_ERROR(hipEventCreate(&stop2));//time counting
	HANDLE_ERROR(hipEventRecord(start2, 0));//time counting

	HANDLE_ERROR(hipMalloc((void**)&dev_cn2_global_ind, ext_cn2 * sizeof(unsigned int)));
	HANDLE_ERROR(hipMemcpy(dev_cn2_global_ind, host_cn2_global_ind, ext_cn2 * sizeof(unsigned int), hipMemcpyHostToDevice));
	//Bitonic sort to extract matches core
	//dev_cn2_error are both input and output, in which output sorted distance errors are rewrited in dev_cn2_error
	//dev_cn2_global_ind is adjoint index output after sorting.
	for (i = 1; i <= temp_dim_m_I_cn24[4]; i++)//dim_m_I_cn24[4]
	{
		for (j = i; j > 0; j--) {
			Bitonic_sort_ex_mc << <blocksPerGrid, threadsPerBlock >> > (dev_cn2_error, dev_cn2_global_ind, i, j);
		}
	}
	hipDeviceSynchronize();

	HANDLE_ERROR(hipMemcpy(host_cn2_error, dev_cn2_error, ext_cn2 * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(host_cc4_core_global_ind, dev_cn2_global_ind, cc4 * sizeof(unsigned int), hipMemcpyDeviceToHost));
	// get stop time, and display the timing results
	HANDLE_ERROR(hipEventRecord(stop2, 0));//time counting
	HANDLE_ERROR(hipEventSynchronize(stop2));//time counting
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime2, start2, stop2));//time counting
	printf("Time for Bitonic sort of two pairs sampling using CUDA after extend cn2 to 2^k ext_cn2: %3.1f ms\n", elapsedTime2);//time counting

																															   //unbind texture variables
	HANDLE_ERROR(hipEventDestroy(start2));
	HANDLE_ERROR(hipEventDestroy(stop2));
	printf("host_cn2_error: %f %f", *(host_cn2_error + ext_cn2 / 20), *(host_cn2_error + ext_cn2 / 10));


	//======================================================Because of error, there are always many outliers are taken as inliers.
	//In order to avoid the mistakes, we explore inliers in matches with small errors.
	//prepar for candidates of the matches for exploration.
	int *host_candi_matches_global_ind, n_candidate, *candi_matches, card_candi = 0;
	n_candidate = cc4;
	printf("cn2:%d, ext_cn2:%d, n_candidate:%d\n", cn2, ext_cn2, n_candidate);
	host_candi_matches_global_ind = (int *)malloc(n_candidate * sizeof(unsigned int));
	candi_matches = (int *)malloc(n_matches * sizeof(unsigned int));
	HANDLE_ERROR(hipMemcpy(host_candi_matches_global_ind, dev_cn2_global_ind, n_candidate * sizeof(unsigned int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(dev_cn2_error));
	HANDLE_ERROR(hipFree(dev_cn2_global_ind));

	thrust::host_vector<int> matches_indx(n_matches, 0);
	for (i = 0; i < n_candidate; i++)
	{
		matches_indx[host_ij_in_ind[*(host_candi_matches_global_ind + i)].x] += 1;
		matches_indx[host_ij_in_ind[*(host_candi_matches_global_ind + i)].y] += 1;
	}
	for (i = 0; i < n_matches; i++)
	{
		if (matches_indx[i]>1)//default set to explore core with not less than 2 occurrences
		{
			//printf("matches_indx[i]: %d", matches_indx[i]);
			card_candi++;
			*(candi_matches + i) = 1;
			if (card_candi > n_candidate)
				break;
		}
		else
			*(candi_matches + i) = 0;

	}
	printf("card_candi: %d ", card_candi);



	//================================================Produce c(n,4) combination list
	//Preparing: In order to produce c(n,4) combination list, we first need to transform global index of core to i,j index of core in matches.
	//core_ij_indx(i)=1 indicate the ith match pair is core match;
	thrust::host_vector<int> core_ij_indx(n_matches, 0);
	for (i = 0; i < cc4; i++)
	{
		//printf("*(host_ij_in_ind+*(host_cc4_core_global_ind + i))=%d ", host_ij_in_ind [*(host_cc4_core_global_ind + i)].x);
		//host_cc4_core_global_ind is intercepted c(n,4) global index corresponds to top min error e(matchi,matchj).
		core_ij_indx[host_ij_in_ind[*(host_cc4_core_global_ind + i)].x] += 1;
		core_ij_indx[host_ij_in_ind[*(host_cc4_core_global_ind + i)].y] += 1;
	}
	//count cardinal number of core: card(core_ij_indx)
	int card_core = 0, core_upper_lim = 300, occur_thresh=1;
	for (i = 0; i < n_matches; i++)
	{
		if (core_ij_indx[i]>occur_thresh)//default set to explore core with not less than 2 occurrences
		{
			//			printf("i: %d, core_ij_indx[i]: %d\n",i, core_ij_indx[i]);
			card_core++;
			if (card_core > core_upper_lim)
				break;
		}
	}

	printf("\card_core=%d\n", card_core);

	//thrust::host_vector<int> cm_ind(card_core);
	int *cm_ind, *dev_cm_ind;
	cm_ind = (int *)malloc(card_core * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**)&dev_cm_ind, card_core * sizeof(int)));
	j = 0;
	//	fp = fopen("core_list.txt", "w");
	if (temp)
	{
		for (i = 0; i < n_matches; i++)
			if (core_ij_indx[i])
			{
				cm_ind[j++] = i;
				if (j > core_upper_lim)//according to the core limitation
					break;
			}
	}
	else//explore core with default set
		for (i = 0; i < n_matches; i++)
			if (core_ij_indx[i] > occur_thresh)
			{
				cm_ind[j++] = i;
				if (j > core_upper_lim)//according to the core limitation
					break;
			}
	HANDLE_ERROR(hipMemcpy(dev_cm_ind, cm_ind, card_core * sizeof(int), hipMemcpyHostToDevice));


	blocksPerGrid = device_prop.multiProcessorCount << 1;
	//	threadsPerBlock = device_prop.maxThreadsPerMultiProcessor;
	threadsPerBlock = 256;//256;//based the number of variables used in CUDA __global__ function (total number of registers per block supported by 1080 is 64k)
	hipEvent_t start5, stop5;	//time counting
	float elapsedTime5;//time counting
	HANDLE_ERROR(hipEventCreate(&start5));//time counting
	HANDLE_ERROR(hipEventCreate(&stop5));//time counting
	HANDLE_ERROR(hipEventRecord(start5, 0));//time counting

	int s_d_check = 0, n_round = 400;//100 every GPU thread run n_round to return a result to host
	double q, eps = 1e-30, epsilon = 1e-6, temp_double;
	n_round = card_core * p_core;
	temp_double = card_core / n_round  * (card_core - 1) / (n_round-1) * (card_core - 2) / (n_round - 2) * (card_core - 3) / (n_round - 3) * (card_core - 4) / (n_round - 4) * (card_core - 5) / (n_round - 5) * (card_core - 6) / (n_round - 6);
	n_round = temp_double * 2 / threadsPerBlock / blocksPerGrid;

	printf("\nEvery GPU thread run %d round to return a result to host\n", n_round);
	unsigned int T_iter = 10000, max_iter = 100000, iter = 0;
	int temp_count = 0, N_I_star = 0, *dev_result_ninliers, *host_result_ninliers, result_ninliers, temp_results;
	float *nrmlz_frames0x, *nrmlz_frames0y, *nrmlz_frames1x, *nrmlz_frames1y, *dev_nrmlz_frames0x, *dev_nrmlz_frames1x, *dev_nrmlz_frames0y, *dev_nrmlz_frames1y;
	;
	double T0[ndof], T1[ndof], *dev_T0, *dev_T1;
	nrmlz_frames0x = (float *)malloc(n_matches * sizeof(float));
	nrmlz_frames0y = (float *)malloc(n_matches * sizeof(float));
	nrmlz_frames1x = (float *)malloc(n_matches * sizeof(float));
	nrmlz_frames1y = (float *)malloc(n_matches * sizeof(float));

	int *dev_result_MSS, *host_result_MSS, results_MSS[fund_fd];
	HANDLE_ERROR(hipMalloc((void**)&dev_result_ninliers, blocksPerGrid * sizeof(int)));//GPU write results back to global memory, since every block works independently in their shared memory.
	host_result_ninliers = (int *)malloc(blocksPerGrid * sizeof(int));
	for (i = 0; i<blocksPerGrid; i++)
		host_result_ninliers[i] = 0;
	HANDLE_ERROR(hipMemcpy(dev_result_ninliers, host_result_ninliers, blocksPerGrid * sizeof(int), hipMemcpyHostToDevice));

	//we design a variable with the length of the number of blocks to deliver results back to host memory
	HANDLE_ERROR(hipMalloc((void**)&dev_result_MSS, blocksPerGrid * 8 * sizeof(int)));
	host_result_MSS = (int *)malloc(blocksPerGrid * 8 * sizeof(int));

	result_ninliers = 0;//we use it to collect the results in array host_result_MSS
	for (i = 0; i < fund_fd; i++)
		results_MSS[i] = 0;
	//For a minimal sample list with a length l_MSL, draw a subset with the length threadsPerBlock * blocksPerGrid * n_round and store to variable rand_ind

	//If RANSAC is implemented on a fundamental matrix, which is conducted by epipolar geometry model, all the coordinates of the matches should be normalized in the interval [-sqrt(2), sqrt(2)]
	//As for RANSAC for homography, the normalization is only implemented on the random hit 4 matches, and we implement the normalization in RANSAC_homog
	normal_samples_fund(n_matches, frames0x, frames0y, nrmlz_frames0x, nrmlz_frames0y, T0);
	normal_samples_fund(n_matches, frames1x, frames1y, nrmlz_frames1x, nrmlz_frames1y, T1);
	HANDLE_ERROR(hipMalloc((void**)&dev_T0, ndof * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T1, ndof * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_nrmlz_frames0x, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_nrmlz_frames1x, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_nrmlz_frames0y, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_nrmlz_frames1y, n_matches * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_T0, T0, ndof * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_T1, T1, ndof * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_nrmlz_frames0x, nrmlz_frames0x, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_nrmlz_frames1x, nrmlz_frames1x, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_nrmlz_frames0y, nrmlz_frames0y, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_nrmlz_frames1y, nrmlz_frames1y, n_matches * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_n_round), &n_round, sizeof(int)));
	for (i = 0; i < fund_fd; i++)
		s_d_check += i;

	while (iter <= T_iter && iter <= max_iter)
	{	//Input: n_matches, dev_frames0x,...
		//Output:dev_results.x is a vector with the length gridDim, in which the i-th element is the maximum number of inliers in the i-th block.
		//dev_results.y is a vector with the length gridDim. The i-th element is the index in 4 samples list which implied maximum inliers in the i-th block.
		//__global__ void RANSAC_fund(float *dev_frames0x, float *dev_frames0y, float *dev_frames1x, float *dev_frames1y, int l_list, int s_d_check, int *cm_ind, int card_core, int *result_ninliers, int *result_MSS)

		RANSAC_fund << <blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int) * 8 >> > (dev_nrmlz_frames0x, dev_nrmlz_frames0y, dev_nrmlz_frames1x, dev_nrmlz_frames1y, dev_frames0x, dev_frames0y, dev_frames1x, dev_frames1y, dev_T0, dev_T1, n_matches, s_d_check, dev_cm_ind, card_core, dev_result_ninliers, dev_result_MSS);
		HANDLE_ERROR(hipMemcpy(host_result_ninliers, dev_result_ninliers, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(host_result_MSS, dev_result_MSS, blocksPerGrid * 8 * sizeof(int), hipMemcpyDeviceToHost));

		//Find optimal results 
		result_ninliers = host_result_ninliers[0];//Initialization
												  //printf("\nhost_result_MSS[j]\n");
		for (j = 0; j < fund_fd; j++)
		{
			results_MSS[j] = host_result_MSS[j];
			//printf("%d ", host_result_MSS[j]);
		}

		for (i = 1; i < blocksPerGrid; i++)
		{
			if (host_result_ninliers[i] > result_ninliers)
			{
				result_ninliers = host_result_ninliers[i];
				for (j = 0; j<fund_fd; j++)
					results_MSS[j] = host_result_MSS[i*fund_fd + j];
			}
		}
		printf("current host loop: %d, find %d inliers, corresponding optimal MSS: %d %d %d  %d %d %d %d\n", iter, result_ninliers, results_MSS[0], results_MSS[1], results_MSS[2], results_MSS[3], results_MSS[4], results_MSS[5], results_MSS[6]);
		temp_count = result_ninliers;
		if (temp_count > N_I_star)
		{
			N_I_star = temp_count;

			//update the number of iterations
			q = (double)(N_I_star) / (double)(n_matches);
			temp_double = 1 - q*q*q*q*q*q*q;
			//			printf("N_I_star: %d, n_matches:%d, q:%f, temp:%f\n", N_I_star, n_matches,q, temp_double);
			q = eps > temp_double ? eps : temp_double;
			temp_double = 1 - eps < q ? 1 - eps : q;
			//			printf("q>eps:%d, %5.16f, temp: %5.16f\n", q > eps, q, temp_double);
			T_iter = (unsigned int)(log(0.01) / log(temp_double));
			//			if (T_iter < 0)
			//				printf("T_iter=%d,log(epsilon)=%f,log(1 - q)=%f,T_iter=%ld\n", T_iter, log(epsilon), log(1 - q), round(log(epsilon) / log(1 - q)));
			printf("\nupdate the RANSAC results, total iter need: %u, n_inliers: %d, MSS: %d, %d, %d, %d, %d, %d, %d\n", T_iter, N_I_star, results_MSS[0], results_MSS[1], results_MSS[2], results_MSS[3], results_MSS[4], results_MSS[5], results_MSS[6]);
		}
		if (temp_count == n_matches)
			break;
		//the main loop contains every thread runs n_round RANSAC exploring to return results to host memory.
		iter = iter + threadsPerBlock*blocksPerGrid*n_round;
		printf("iter: %d\n", iter);
	}
	//=======================================================================================


	HANDLE_ERROR(hipEventRecord(stop5, 0));//time counting
	HANDLE_ERROR(hipEventSynchronize(stop5));//time counting
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime5, start5, stop5));//time counting
	printf("Time for RANSAC against core list: %3.1f ms\n", elapsedTime5);//time counting
	printf("\nTime for GPU processing totals to: %3.1f ms\n", elapsedTime1 + elapsedTime2 + elapsedTime5);//time counting

	HANDLE_ERROR(hipFree(dev_frames0x));
	HANDLE_ERROR(hipFree(dev_frames0y));
	HANDLE_ERROR(hipFree(dev_frames1x));
	HANDLE_ERROR(hipFree(dev_frames1y));
	HANDLE_ERROR(hipFree(dev_result_MSS));
	HANDLE_ERROR(hipFree(dev_result_ninliers));


	if (result_ninliers == 0)
	{
		printf("There is no inlier explored. \n");
		exit(0);
	}

	int rank_results = 0;
	//in some case, different MSS return a same number of inliers, and we need to explore every MSS with a same maximum number of inliers
	for (i = 0; i < blocksPerGrid; i++)
		if (host_result_ninliers[i] == N_I_star)//result_ninliers
			rank_results++;

	int *results_array = (int*)calloc(rank_results, sizeof(int));
	j = 0;
	for (i = 0; i < blocksPerGrid; i++)
	{
		if (host_result_ninliers[i] == N_I_star)//result_ninliers
		{
			*(results_array + j) = i;
			j++;
		}
	}

	double F[N3][N3], F_star[N3][N3],  proj_error = 0.0;
	int *CS, *CS_temp, *CS_outlieres, *CS_outlieres_temp;
	int ranki, n_temp = 0, ninliers = 0, k = 0, n_temp_init, n_eliminated, non_degenerated = 0;

	F[0][0] = 0.0; F[0][1] = 0.0; F[0][2] = 0.0;
	F[1][0] = 0.0; F[1][1] = 0.0; F[1][2] = 0.0;
	F[2][0] = 0.0; F[2][1] = 0.0; F[2][2] = 0.0;

	CS = (int*)calloc(n_matches, sizeof(int));
	CS_temp = (int*)calloc(n_matches, sizeof(int));
	CS_outlieres = (int*)calloc(n_matches, sizeof(int));
	CS_outlieres_temp = (int*)calloc(n_matches, sizeof(int));

	printf("\nrank_results: %d\n", rank_results);
	for (ranki = 0; ranki < rank_results; ranki++)
	{//host_result_MSS: point to the head of an 7 points MSS
//output:
//CS_temp: if CS_temp[i]=1, the i-th correpondence is a true correspondence
//function value: n_temp is the number of inliers evaluated with the model modeled by the MSS host_result_MSS
		n_temp = assembleinliers(host_result_MSS + *(results_array + ranki)*fund_fd, nrmlz_frames0x, nrmlz_frames0y, nrmlz_frames1x, nrmlz_frames1y, frames0x, frames0y, frames1x, frames1y, T0, T1, host_cn2_error_beforesort, n_matches, CS_temp);
		n_temp_init = n_temp;

		for (j = 0; j < n_matches; j++)
			*(CS_outlieres_temp + j) = 0;
		for (j = 0; j < 2; j++)//two runs to eliminate outliers more thoroughly
		{	//output: CS_temp that eliminated outliers
			//CS_outlieres_temp that indicate outilers
			n_eliminated = outliers_detec(frames0x, frames0y, frames1x, frames1y, n_matches, n_temp, CS_temp, CS_outlieres_temp);
			n_temp -= n_eliminated;
		}

		//fitting a model F using the inliers in *CS_temp, where *(CS_temp+i)=1 indicates an inlier, n_temp is the total inliers
		//Output: F
		if (n_temp > 7)
		{
			reestimate(frames0x, frames0y, frames1x, frames1y, n_matches, n_temp, CS_temp, F_star);
			non_degenerated = 1;
		}
		//////////////////////////////////////////////////////////////////////

		if (n_temp > ninliers & non_degenerated)
		{
			ninliers = n_temp;
			for (j = 0; j < N3; j++)
				for (k = 0; k < N3; k++)
				{
					F[j][k] = F_star[j][k];
					printf("%f ", F[j][k]);
				}
			for (j = 0; j < n_matches; j++)
			{
				*(CS_outlieres + j) = *(CS_outlieres_temp + j);
				*(CS + j) = *(CS_temp + j);
			}
		}
	}
	printf("\nexplored the number of inliers: %d\n", ninliers);
	strcat(file_F, filename);
	fp = fopen(file_F, "w");
	for (i = 0; i < N3; i++)
	{
		for (j = 0; j < N3; j++)
		{
			printf("%5.12f ", F[i][j]);
			fprintf(fp, "%5.12f  ", F[i][j]);
		}
		printf("\n");
		fprintf(fp, "\n");

	}
	fclose(fp);

	if (non_degenerated)
	{	//estimate the projection error 
		proj_error = est_proj_error(frames0x, frames0y, frames1x, frames1y, n_matches, CS, CS_outlieres, &N_I_star, F);
		printf("proj_error: %f", proj_error);

		strcat(outliersstr, filename);
		fp = fopen(outliersstr, "w");
		for (i = 0; i < n_matches; i++)
			if (*(CS_outlieres + i))
				fprintf(fp, "%d ", i);
		fclose(fp);

		ninliers = 0;
		strcat(file_a, filename);
		fp = fopen(file_a, "w");
		for (i = 0; i < n_matches; i++)
			if (*(CS + i))
			{
				fprintf(fp, "%d ", i);
				ninliers++;
			}
		fclose(fp);
		printf("\nFinally explored the number of inliers satisfying CSAC constraints: %d\n", ninliers);
		ninliers = N_I_star;

		QueryPerformanceCounter(&time_over);    //��ʱ����  
		duration = 1000 * (time_over.QuadPart - time_start.QuadPart) / dqFreq;
		printf("\nThe time cost against the whole demo, including GPU and CPU processing time (ms): %3.2f\n", duration);

		strcat(file_abstraction, filename);
		fp = fopen(file_abstraction, "w");
		////total number of correspondences|card_core| ninliers before reestimate | ninliers| cn2| Bitonic sort| RANSAC against core list| GPU| the whole time cost| proj_error | cardb (cosi in Eq. (38)) | Tlb
		fprintf(fp, "%d\t %d\t %d\t %d\t %d\t %f\t %f\t %f\t %f\t %f\t %f\t", n_matches, card_core, n_round, n_temp_init, ninliers, elapsedTime1, elapsedTime2, elapsedTime5, elapsedTime1 + elapsedTime2 + elapsedTime5, duration, proj_error);
		fclose(fp);

	}
	else
	{
		strcat(file_abstraction, filename);
		fp = fopen(file_abstraction, "w");
		//total number of correspondences| card_core| ninliers before n-1/n inliers model fitting | ninliers| cn2| Bitonic sort| RANSAC against core list| GPU| the whole time cost| proj_error | cardb (cosi in Eq. (38)) | Tlb
		fprintf(fp, "%d\t %d\t %d\t %d\t %d\t %f\t %f\t %f\t %f\t %f\t", n_matches, card_core, n_round, n_temp_init, ninliers, elapsedTime1, elapsedTime2, elapsedTime5, elapsedTime1 + elapsedTime2 + elapsedTime5, elapsedTime1 + elapsedTime2 + elapsedTime5);
		fclose(fp);
	}

	free(results_array);
	free(CS);
	free(matches1);
	free(matches2);
	free(I0);
	free(I1);
	free(host_cn2_error);
	free(host_cn2_error_beforesort);
	free(host_cn2_global_ind);
	free(host_ij_in_ind);
	free(host_cc4_core_global_ind);
	free(host_candi_matches_global_ind);
	free(candi_matches);
	free(cm_ind);
	free(nrmlz_frames0x);
	free(nrmlz_frames0y);
	free(nrmlz_frames1x);
	free(nrmlz_frames1y);
	free(host_result_ninliers);
	free(host_result_MSS);

}